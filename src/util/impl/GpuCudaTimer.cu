#ifndef _GPU_CUDA_TIMER_CU__
#define _GPU_CUDA_TIMER_CU__

#include <stdio.h>
#include <stdlib.h>
#include "../../util/GpuCudaTimer.h"
#include "../../util/GpuCudaHelper.h"

namespace SiddhiGpu
{

GpuCudaTimer::GpuCudaTimer()
{
	CUDA_CHECK_RETURN(hipEventCreate(&start));
	CUDA_CHECK_RETURN(hipEventCreate(&end));
	CUDA_CHECK_RETURN(hipEventRecord(start,0));
}

GpuCudaTimer::~GpuCudaTimer()
{
	CUDA_CHECK_RETURN(hipEventDestroy(start));
	CUDA_CHECK_RETURN(hipEventDestroy(end));
}

float GpuCudaTimer::MillisecondsElapsed()
{
	float elapsed_time;
	CUDA_CHECK_RETURN(hipEventRecord(end, 0));
	CUDA_CHECK_RETURN(hipEventSynchronize(end));
	CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsed_time, start, end));
	return elapsed_time;
}

float GpuCudaTimer::SecondsElapsed()
{
	return 1000.0 * MillisecondsElapsed();
}

}


#endif
