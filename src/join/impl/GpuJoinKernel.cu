#include "hip/hip_runtime.h"
#ifndef _GPU_JOIN_KERNEL_CU__
#define _GPU_JOIN_KERNEL_CU__

#include <stdio.h>
#include <stdlib.h>
#include "../../domain/GpuMetaEvent.h"
#include "../../main/GpuProcessor.h"
#include "../../domain/GpuProcessorContext.h"
#include "../../buffer/GpuStreamEventBuffer.h"
#include "../../buffer/GpuWindowEventBuffer.h"
#include "../../buffer/GpuRawByteBuffer.h"
#include "../../buffer/GpuIntBuffer.h"
#include "../../domain/GpuKernelDataTypes.h"
#include "../../join/GpuJoinProcessor.h"
#include "../../join/GpuJoinKernel.h"
#include "../../util/GpuCudaHelper.h"
#include "../../join/GpuJoinKernelCore.h"
#include "../../filter/GpuFilterProcessor.h"
#include "../../util/GpuUtils.h"

namespace SiddhiGpu
{

#define THREADS_PER_BLOCK 128
#define MY_KERNEL_MAX_THREADS THREADS_PER_BLOCK
#define MY_KERNEL_MIN_BLOCKS 8

// process batch of events in one stream of join processor
__global__
void
__launch_bounds__(MY_KERNEL_MAX_THREADS, MY_KERNEL_MIN_BLOCKS)
ProcessEventsJoinLeftTriggerAllOn(
		char               * _pInputEventBuffer,         // input events buffer
		GpuKernelMetaEvent * _pInputMetaEvent,           // Meta event for input events
		int                  _iInputNumberOfEvents,      // Number of events in input buffer
		char               * _pEventWindowBuffer,        // Event window buffer of this stream
		int                  _iWindowLength,             // Length of current events window
		int                  _iRemainingCount,           // Remaining free slots in Window buffer
		GpuKernelMetaEvent * _pOtherStreamMetaEvent,     // Meta event for other stream
		char               * _pOtherEventWindowBuffer,   // Event window buffer of other stream
		int                  _iOtherWindowLength,        // Length of current events window of other stream
		int                  _iOtherRemainingCount,      // Remaining free slots in Window buffer of other stream
		GpuKernelFilter    * _pOnCompareFilter,          // OnCompare filter buffer - pre-copied at initialization
		uint64_t             _iWithInTime,               // WithIn time in milliseconds
		GpuKernelMetaEvent * _pOutputStreamMetaEvent,    // Meta event for output stream
		char               * _pResultsBuffer,            // Resulting events buffer for this stream
		AttributeMappings  * _pOutputAttribMappings,     // Output event attribute mappings
		int                  _iEventsPerBlock            // number of events allocated per block
)
{
	// avoid out of bound threads
	if(threadIdx.x >= _iEventsPerBlock || threadIdx.y > 0 || blockIdx.y > 0)
		return;

	if((blockIdx.x == _iInputNumberOfEvents / _iEventsPerBlock) && // last thread block
			(threadIdx.x >= _iInputNumberOfEvents % _iEventsPerBlock)) // extra threads
	{
		return;
	}

	// get assigned event
	int iEventIdx = (blockIdx.x * _iEventsPerBlock) + threadIdx.x;

	// get in event starting position
	char * pInEventBuffer = _pInputEventBuffer + (_pInputMetaEvent->i_SizeOfEventInBytes * iEventIdx);

	// output to results buffer [in event, expired event]
	// {other stream event size * other window size} * 2 (for in/exp)
	int iOutputSegmentSize = _pOutputStreamMetaEvent->i_SizeOfEventInBytes * _iOtherWindowLength * 2;

	char * pResultsInEventBufferSegment = _pResultsBuffer + (iOutputSegmentSize * iEventIdx);
	char * pResultsExpiredEventBufferSegment = pResultsInEventBufferSegment + (iOutputSegmentSize / 2);

	char * pExpiredEventBuffer = NULL;
	GpuEvent * pExpiredEvent = NULL;

	GpuEvent * pInEvent = (GpuEvent*) pInEventBuffer;

	// calculate in/expired event pair for this event

	if(iEventIdx >= _iRemainingCount)
	{
		if(iEventIdx < _iWindowLength)
		{
			// in window buffer
			char * pExpiredOutEventInWindowBuffer = _pEventWindowBuffer + (_pInputMetaEvent->i_SizeOfEventInBytes * (iEventIdx - _iRemainingCount));

			GpuEvent * pWindowEvent = (GpuEvent*) pExpiredOutEventInWindowBuffer;
			if(pWindowEvent->i_Type != GpuEvent::NONE) // if window event is filled
			{
				pExpiredEventBuffer = pExpiredOutEventInWindowBuffer;

			}
			else
			{
				// no expiring event
			}
		}
		else
		{
			// in input event buffer
			char * pExpiredOutEventInInputBuffer = _pInputEventBuffer + (_pInputMetaEvent->i_SizeOfEventInBytes * (iEventIdx - _iWindowLength));
			pExpiredEventBuffer = pExpiredOutEventInInputBuffer;
		}
	}
	else
	{
		// [NULL,inEvent]
		// no expiring event
	}


	// get all matching event for in event from other window buffer and copy them to output event buffer

	// for each events in other window
	int iOtherWindowFillCount  = _iOtherWindowLength - _iOtherRemainingCount;
	int iMatchedCount = 0;
	for(int i=0; i<iOtherWindowFillCount; ++i)
	{
		// get other window event
		char * pOtherWindowEventBuffer = _pOtherEventWindowBuffer + (_pOtherStreamMetaEvent->i_SizeOfEventInBytes * i);
		GpuEvent * pOtherWindowEvent = (GpuEvent*) pOtherWindowEventBuffer;

		// get buffer position for in event matching results
		char * pResultInMatchingEventBuffer = pResultsInEventBufferSegment + (_pOutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
		GpuEvent * pResultInMatchingEvent = (GpuEvent*) pResultInMatchingEventBuffer;

		if(pInEvent->i_Sequence > pOtherWindowEvent->i_Sequence &&
				(pInEvent->i_Timestamp - pOtherWindowEvent->i_Timestamp) <= _iWithInTime)
		{
			ExpressionEvalParameters mExpressionParam;
			mExpressionParam.p_OnCompare = _pOnCompareFilter;
			mExpressionParam.a_Meta[0] = _pInputMetaEvent;
			mExpressionParam.a_Event[0] = pInEventBuffer;
			mExpressionParam.a_Meta[1] = _pOtherStreamMetaEvent;
			mExpressionParam.a_Event[1] = pOtherWindowEventBuffer;
			mExpressionParam.i_CurrentIndex = 0;

			bool bOnCompareMatched = Evaluate(mExpressionParam);
			if(bOnCompareMatched)
			{
				// copy output event to buffer - map attributes from input streams to output stream
				pResultInMatchingEvent->i_Type = GpuEvent::CURRENT;
				pResultInMatchingEvent->i_Sequence = pInEvent->i_Sequence;
				pResultInMatchingEvent->i_Timestamp = pInEvent->i_Timestamp;

				for(int m=0; m < _pOutputAttribMappings->i_MappingCount; ++m)
				{
					int iFromStreamIndex = _pOutputAttribMappings->p_Mappings[m].from[AttributeMapping::STREAM_INDEX];
					int iFromAttrib = _pOutputAttribMappings->p_Mappings[m].from[AttributeMapping::ATTRIBUTE_INDEX];
					int iTo = _pOutputAttribMappings->p_Mappings[m].to;

					memcpy(
						pResultInMatchingEventBuffer + _pOutputStreamMetaEvent->p_Attributes[iTo].i_Position, // to
						mExpressionParam.a_Event[iFromStreamIndex] + mExpressionParam.a_Meta[iFromStreamIndex]->p_Attributes[iFromAttrib].i_Position, // from
						mExpressionParam.a_Meta[iFromStreamIndex]->p_Attributes[iFromAttrib].i_Length // size
					);
				}

				iMatchedCount++;
			}
		}
		else
		{
			// cannot continue, last result event for this segment
			pResultInMatchingEvent->i_Type = GpuEvent::RESET;
			break;
		}
	}

	if(iMatchedCount < iOtherWindowFillCount || iOtherWindowFillCount == 0)
	{
		char * pResultInMatchingEventBuffer = pResultsInEventBufferSegment + (_pOutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
		GpuEvent * pResultInMatchingEvent = (GpuEvent*) pResultInMatchingEventBuffer;
		pResultInMatchingEvent->i_Type = GpuEvent::RESET;
	}

	if(pExpiredEventBuffer != NULL)
	{
		pExpiredEvent = (GpuEvent*) pExpiredEventBuffer;

		iMatchedCount = 0;
		// for each events in other window
		for(int i=0; i<iOtherWindowFillCount; ++i)
		{
			// get other window event
			char * pOtherWindowEventBuffer = _pOtherEventWindowBuffer + (_pOtherStreamMetaEvent->i_SizeOfEventInBytes * i);
			GpuEvent * pOtherWindowEvent = (GpuEvent*) pOtherWindowEventBuffer;

			// get buffer position for expire event matching results
			char * pResultExpireMatchingEventBuffer = pResultsExpiredEventBufferSegment + (_pOutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
			GpuEvent * pResultExpireMatchingEvent = (GpuEvent*) pResultExpireMatchingEventBuffer;

			if(pExpiredEvent->i_Sequence < pOtherWindowEvent->i_Sequence &&
					(pOtherWindowEvent->i_Timestamp - pExpiredEvent->i_Timestamp) <= _iWithInTime)
			{
				ExpressionEvalParameters mExpressionParam;
				mExpressionParam.p_OnCompare = _pOnCompareFilter;
				mExpressionParam.a_Meta[0] = _pInputMetaEvent;
				mExpressionParam.a_Event[0] = pExpiredEventBuffer;
				mExpressionParam.a_Meta[1] = _pOtherStreamMetaEvent;
				mExpressionParam.a_Event[1] = pOtherWindowEventBuffer;
				mExpressionParam.i_CurrentIndex = 0;

				bool bOnCompareMatched = Evaluate(mExpressionParam);
				if(bOnCompareMatched)
				{
					// copy output event to buffer - map attributes from input streams to output stream
					pResultExpireMatchingEvent->i_Type = GpuEvent::EXPIRED;
					pResultExpireMatchingEvent->i_Sequence = pExpiredEvent->i_Sequence;
					pResultExpireMatchingEvent->i_Timestamp = pExpiredEvent->i_Timestamp;

					for(int m=0; m < _pOutputAttribMappings->i_MappingCount; ++m)
					{
						int iFromStreamIndex = _pOutputAttribMappings->p_Mappings[m].from[AttributeMapping::STREAM_INDEX];
						int iFromAttrib = _pOutputAttribMappings->p_Mappings[m].from[AttributeMapping::ATTRIBUTE_INDEX];
						int iTo = _pOutputAttribMappings->p_Mappings[m].to;

						memcpy(
								pResultExpireMatchingEventBuffer + _pOutputStreamMetaEvent->p_Attributes[iTo].i_Position, // to
								mExpressionParam.a_Event[iFromStreamIndex] + mExpressionParam.a_Meta[iFromStreamIndex]->p_Attributes[iFromAttrib].i_Position, // from
								mExpressionParam.a_Meta[iFromStreamIndex]->p_Attributes[iFromAttrib].i_Length // size
						);
					}

					iMatchedCount++;
				}
			}
			else
			{
				// cannot continue, last result event for this segment
				pResultExpireMatchingEvent->i_Type = GpuEvent::RESET;
				break;
			}
		}

		if(iMatchedCount < iOtherWindowFillCount || iOtherWindowFillCount == 0)
		{
			char * pResultExpireMatchingEventBuffer = pResultsExpiredEventBufferSegment + (_pOutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
			GpuEvent * pResultExpireMatchingEvent = (GpuEvent*) pResultExpireMatchingEventBuffer;
			pResultExpireMatchingEvent->i_Type = GpuEvent::RESET;
		}
	}

}

//__global__
//void ProcessEventsJoinLeftTriggerCurrentOn(
//		char               * _pInputEventBuffer,         // input events buffer
//		GpuKernelMetaEvent * _pInputMetaEvent,           // Meta event for input events
//		int                  _iInputNumberOfEvents,      // Number of events in input buffer
//		char               * _pEventWindowBuffer,        // Event window buffer of this stream
//		int                  _iWindowLength,             // Length of current events window
//		int                  _iRemainingCount,           // Remaining free slots in Window buffer
//		GpuKernelMetaEvent * _pOtherStreamMetaEvent,     // Meta event for other stream
//		char               * _pOtherEventWindowBuffer,   // Event window buffer of other stream
//		int                  _iOtherWindowLength,        // Length of current events window of other stream
//		int                  _iOtherRemainingCount,      // Remaining free slots in Window buffer of other stream
//		GpuKernelFilter    * _pOnCompareFilter,          // OnCompare filter buffer - pre-copied at initialization
//		uint64_t             _iWithInTime,               // WithIn time in milliseconds
//		GpuKernelMetaEvent * _pOutputStreamMetaEvent,    // Meta event for output stream
//		char               * _pResultsBuffer,            // Resulting events buffer for this stream
//		AttributeMappings  * _pOutputAttribMappings,     // Output event attribute mappings
//		int                  _iEventsPerBlock,           // number of events allocated per block
//		int                  _iWorkSize                  // Number of events in window process by this kernel
//)

__global__
void
__launch_bounds__(MY_KERNEL_MAX_THREADS, MY_KERNEL_MIN_BLOCKS)
ProcessEventsJoinLeftTriggerCurrentOn(
		JoinKernelParameters * _pParameters,
		int                   _iInputNumberOfEvents,      // Number of events in input buffer
		int                   _iRemainingCount,           // Remaining free slots in Window buffer
		int                   _iOtherRemainingCount      // Remaining free slots in Window buffer of other stream
)
{
	// avoid out of bound threads
	if(threadIdx.x >= _pParameters->i_EventsPerBlock || threadIdx.y > 0 || blockIdx.y > 0)
		return;

	int iWorkerCount = ceil((float)_pParameters->i_OtherWindowLength / _pParameters->i_WorkSize);

	if((blockIdx.x == (_iInputNumberOfEvents * iWorkerCount) / _pParameters->i_EventsPerBlock) && // last thread block
			(threadIdx.x >= (_iInputNumberOfEvents * iWorkerCount) % _pParameters->i_EventsPerBlock)) // extra threads
	{
		return;
	}

	extern __shared__ char p_SharedInputEventBuffer[];

	// get assigned event
	int iGlobalThreadIdx = (blockIdx.x * _pParameters->i_EventsPerBlock) + threadIdx.x;

	// get in buffer index
	int iInEventIndex = iGlobalThreadIdx / iWorkerCount;
	int iWindowStartEventIndex = (iGlobalThreadIdx % iWorkerCount) * _pParameters->i_WorkSize;

	// get in event starting position
//	char * pInEventBuffer = _pParameters->p_InputEventBuffer + (_pParameters->p_InputMetaEvent->i_SizeOfEventInBytes * iInEventIndex);
	char * pSharedInEventBuffer = p_SharedInputEventBuffer + (_pParameters->p_InputMetaEvent->i_SizeOfEventInBytes * (threadIdx.x / iWorkerCount));
	if(threadIdx.x % iWorkerCount == 0)
	{
		memcpy(pSharedInEventBuffer,
			_pParameters->p_InputEventBuffer + (_pParameters->p_InputMetaEvent->i_SizeOfEventInBytes * iInEventIndex),
			_pParameters->p_InputMetaEvent->i_SizeOfEventInBytes);
	}
	__syncthreads();

	// output to results buffer [in event, expired event]
	// {other stream event size * other window size} * 2 (for in/exp)
//	int iOutputSegmentSize = _pOutputStreamMetaEvent->i_SizeOfEventInBytes * _iOtherWindowLength;

	char * pResultsInEventBufferSegment = _pParameters->p_ResultsBuffer + (iGlobalThreadIdx * _pParameters->i_WorkSize * _pParameters->p_OutputStreamMetaEvent->i_SizeOfEventInBytes);
//			+ (iWindowStartEventIndex * _pOutputStreamMetaEvent->i_SizeOfEventInBytes);

	GpuEvent * pInEvent = (GpuEvent*) pSharedInEventBuffer;

//	memset(pResultsInEventBufferSegment, 0, _pParameters->i_WorkSize * _pParameters->p_OutputStreamMetaEvent->i_SizeOfEventInBytes);

	// get all matching event for in event from other window buffer and copy them to output event buffer

	// for each events in other window
	int iOtherWindowFillCount  = _pParameters->i_OtherWindowLength - _iOtherRemainingCount;

	if(iWindowStartEventIndex < iOtherWindowFillCount)
	{
		int iWindowEndEventIndex = min(iWindowStartEventIndex + _pParameters->i_WorkSize, iOtherWindowFillCount);

		int iMatchedCount = 0;
		// get buffer position for in event matching results
		char * pResultInMatchingEventBuffer = pResultsInEventBufferSegment + (_pParameters->p_OutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
		GpuEvent * pResultInMatchingEvent = (GpuEvent*) pResultInMatchingEventBuffer;

		for(int i=iWindowStartEventIndex; i<iWindowEndEventIndex; ++i)
		{
			// get other window event
			char * pOtherWindowEventBuffer = _pParameters->p_OtherEventWindowBuffer + (_pParameters->p_OtherStreamMetaEvent->i_SizeOfEventInBytes * i);
			GpuEvent * pOtherWindowEvent = (GpuEvent*) pOtherWindowEventBuffer;


			if(pInEvent->i_Sequence > pOtherWindowEvent->i_Sequence &&
					(pInEvent->i_Timestamp - pOtherWindowEvent->i_Timestamp) <= _pParameters->i_WithInTime)
			{
				ExpressionEvalParameters mExpressionParam;
				mExpressionParam.p_OnCompare = _pParameters->p_OnCompareFilter;
				mExpressionParam.a_Meta[0] = _pParameters->p_InputMetaEvent;
				mExpressionParam.a_Event[0] = pSharedInEventBuffer;
				mExpressionParam.a_Meta[1] = _pParameters->p_OtherStreamMetaEvent;
				mExpressionParam.a_Event[1] = pOtherWindowEventBuffer;
				mExpressionParam.i_CurrentIndex = 0;

				bool bOnCompareMatched = Evaluate(mExpressionParam);
				if(bOnCompareMatched)
				{
					// copy output event to buffer - map attributes from input streams to output stream
					//

					pResultInMatchingEvent->i_Type = GpuEvent::CURRENT;
					pResultInMatchingEvent->i_Sequence = pInEvent->i_Sequence;
					pResultInMatchingEvent->i_Timestamp = pInEvent->i_Timestamp;

					#pragma __unroll__
					for(int m=0; m < _pParameters->p_OutputAttribMappings->i_MappingCount; ++m)
					{
						int iFromStreamIndex = _pParameters->p_OutputAttribMappings->p_Mappings[m].from[AttributeMapping::STREAM_INDEX];
						int iFromAttrib = _pParameters->p_OutputAttribMappings->p_Mappings[m].from[AttributeMapping::ATTRIBUTE_INDEX];
						int iTo = _pParameters->p_OutputAttribMappings->p_Mappings[m].to;

						memcpy(
								pResultInMatchingEventBuffer + _pParameters->p_OutputStreamMetaEvent->p_Attributes[iTo].i_Position, // to
								mExpressionParam.a_Event[iFromStreamIndex] + mExpressionParam.a_Meta[iFromStreamIndex]->p_Attributes[iFromAttrib].i_Position, // from
								mExpressionParam.a_Meta[iFromStreamIndex]->p_Attributes[iFromAttrib].i_Length // size
						);
					}

					iMatchedCount++;

					pResultInMatchingEventBuffer = pResultsInEventBufferSegment + (_pParameters->p_OutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
					pResultInMatchingEvent = (GpuEvent*) pResultInMatchingEventBuffer;
				}
			}
			else
			{
				pResultInMatchingEventBuffer = pResultsInEventBufferSegment + (_pParameters->p_OutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
				pResultInMatchingEvent = (GpuEvent*) pResultInMatchingEventBuffer;

				// cannot continue, last result event for this segment
				pResultInMatchingEvent->i_Type = GpuEvent::RESET;
				break;
			}
		}

		if(iMatchedCount < (iWindowEndEventIndex - iWindowStartEventIndex))
		{
			char * pResultInMatchingEventBuffer = pResultsInEventBufferSegment + (_pParameters->p_OutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
			GpuEvent * pResultInMatchingEvent = (GpuEvent*) pResultInMatchingEventBuffer;
			pResultInMatchingEvent->i_Type = GpuEvent::RESET;
		}

	}
	else
	{
		GpuEvent * pResultInMatchingEvent = (GpuEvent*) pResultsInEventBufferSegment;
		pResultInMatchingEvent->i_Type = GpuEvent::RESET;
	}
}

__global__
void
__launch_bounds__(MY_KERNEL_MAX_THREADS, MY_KERNEL_MIN_BLOCKS)
ProcessEventsJoinLeftTriggerExpiredOn(
		char               * _pInputEventBuffer,         // input events buffer
		GpuKernelMetaEvent * _pInputMetaEvent,           // Meta event for input events
		int                  _iInputNumberOfEvents,      // Number of events in input buffer
		char               * _pEventWindowBuffer,        // Event window buffer of this stream
		int                  _iWindowLength,             // Length of current events window
		int                  _iRemainingCount,           // Remaining free slots in Window buffer
		GpuKernelMetaEvent * _pOtherStreamMetaEvent,     // Meta event for other stream
		char               * _pOtherEventWindowBuffer,   // Event window buffer of other stream
		int                  _iOtherWindowLength,        // Length of current events window of other stream
		int                  _iOtherRemainingCount,      // Remaining free slots in Window buffer of other stream
		GpuKernelFilter    * _pOnCompareFilter,          // OnCompare filter buffer - pre-copied at initialization
		uint64_t             _iWithInTime,               // WithIn time in milliseconds
		GpuKernelMetaEvent * _pOutputStreamMetaEvent,    // Meta event for output stream
		char               * _pResultsBuffer,            // Resulting events buffer for this stream
		AttributeMappings  * _pOutputAttribMappings,     // Output event attribute mappings
		int                  _iEventsPerBlock            // number of events allocated per block
)
{
	// avoid out of bound threads
	if(threadIdx.x >= _iEventsPerBlock || threadIdx.y > 0 || blockIdx.y > 0)
		return;

	if((blockIdx.x == _iInputNumberOfEvents / _iEventsPerBlock) && // last thread block
			(threadIdx.x >= _iInputNumberOfEvents % _iEventsPerBlock)) // extra threads
	{
		return;
	}

	// get assigned event
	int iEventIdx = (blockIdx.x * _iEventsPerBlock) + threadIdx.x;

	// output to results buffer [in event, expired event]
	// {other stream event size * other window size} * 2 (for in/exp)
	int iOutputSegmentSize = _pOutputStreamMetaEvent->i_SizeOfEventInBytes * _iOtherWindowLength;

	char * pResultsExpiredEventBufferSegment = _pResultsBuffer + (iOutputSegmentSize * iEventIdx);

	char * pExpiredEventBuffer = NULL;
	GpuEvent * pExpiredEvent = NULL;

	// calculate in/expired event pair for this event

	if(iEventIdx >= _iRemainingCount)
	{
		if(iEventIdx < _iWindowLength)
		{
			// in window buffer
			char * pExpiredOutEventInWindowBuffer = _pEventWindowBuffer + (_pInputMetaEvent->i_SizeOfEventInBytes * (iEventIdx - _iRemainingCount));

			GpuEvent * pWindowEvent = (GpuEvent*) pExpiredOutEventInWindowBuffer;
			if(pWindowEvent->i_Type != GpuEvent::NONE) // if window event is filled
			{
				pExpiredEventBuffer = pExpiredOutEventInWindowBuffer;

			}
			else
			{
				// no expiring event
			}
		}
		else
		{
			// in input event buffer
			char * pExpiredOutEventInInputBuffer = _pInputEventBuffer + (_pInputMetaEvent->i_SizeOfEventInBytes * (iEventIdx - _iWindowLength));
			pExpiredEventBuffer = pExpiredOutEventInInputBuffer;
		}
	}
	else
	{
		// [NULL,inEvent]
		// no expiring event
	}

	if(pExpiredEventBuffer != NULL)
	{
		pExpiredEvent = (GpuEvent*) pExpiredEventBuffer;

		// for each events in other window
		//	 get all matching event for in event from other window buffer and copy them to output event buffer
		int iOtherWindowFillCount  = _iOtherWindowLength - _iOtherRemainingCount;
		int iMatchedCount = 0;
		// for each events in other window
		for(int i=0; i<iOtherWindowFillCount; ++i)
		{
			// get other window event
			char * pOtherWindowEventBuffer = _pOtherEventWindowBuffer + (_pOtherStreamMetaEvent->i_SizeOfEventInBytes * i);
			GpuEvent * pOtherWindowEvent = (GpuEvent*) pOtherWindowEventBuffer;

			// get buffer position for expire event matching results
			char * pResultExpireMatchingEventBuffer = pResultsExpiredEventBufferSegment + (_pOutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
			GpuEvent * pResultExpireMatchingEvent = (GpuEvent*) pResultExpireMatchingEventBuffer;

			if(pExpiredEvent->i_Sequence < pOtherWindowEvent->i_Sequence &&
					(pOtherWindowEvent->i_Timestamp - pExpiredEvent->i_Timestamp) <= _iWithInTime)
			{
				ExpressionEvalParameters mExpressionParam;
				mExpressionParam.p_OnCompare = _pOnCompareFilter;
				mExpressionParam.a_Meta[0] = _pInputMetaEvent;
				mExpressionParam.a_Event[0] = pExpiredEventBuffer;
				mExpressionParam.a_Meta[1] = _pOtherStreamMetaEvent;
				mExpressionParam.a_Event[1] = pOtherWindowEventBuffer;
				mExpressionParam.i_CurrentIndex = 0;

				bool bOnCompareMatched = Evaluate(mExpressionParam);
				if(bOnCompareMatched)
				{
					// copy output event to buffer - map attributes from input streams to output stream
					pResultExpireMatchingEvent->i_Type = GpuEvent::EXPIRED;
					pResultExpireMatchingEvent->i_Sequence = pExpiredEvent->i_Sequence;
					pResultExpireMatchingEvent->i_Timestamp = pExpiredEvent->i_Timestamp;

					for(int m=0; m < _pOutputAttribMappings->i_MappingCount; ++m)
					{
						int iFromStreamIndex = _pOutputAttribMappings->p_Mappings[m].from[AttributeMapping::STREAM_INDEX];
						int iFromAttrib = _pOutputAttribMappings->p_Mappings[m].from[AttributeMapping::ATTRIBUTE_INDEX];
						int iTo = _pOutputAttribMappings->p_Mappings[m].to;

						memcpy(
								pResultExpireMatchingEventBuffer + _pOutputStreamMetaEvent->p_Attributes[iTo].i_Position, // to
								mExpressionParam.a_Event[iFromStreamIndex] + mExpressionParam.a_Meta[iFromStreamIndex]->p_Attributes[iFromAttrib].i_Position, // from
								mExpressionParam.a_Meta[iFromStreamIndex]->p_Attributes[iFromAttrib].i_Length // size
						);
					}

					iMatchedCount++;
				}
			}
			else
			{
				// cannot continue, last result event for this segment
				pResultExpireMatchingEvent->i_Type = GpuEvent::RESET;
				break;
			}
		}

		if(iMatchedCount < iOtherWindowFillCount || iOtherWindowFillCount == 0)
		{
			char * pResultExpireMatchingEventBuffer = pResultsExpiredEventBufferSegment + (_pOutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
			GpuEvent * pResultExpireMatchingEvent = (GpuEvent*) pResultExpireMatchingEventBuffer;
			pResultExpireMatchingEvent->i_Type = GpuEvent::RESET;
		}
	}

}

__global__
void
__launch_bounds__(MY_KERNEL_MAX_THREADS, MY_KERNEL_MIN_BLOCKS)
ProcessEventsJoinRightTriggerAllOn(
		char               * _pInputEventBuffer,         // input events buffer
		GpuKernelMetaEvent * _pInputMetaEvent,           // Meta event for input events
		int                  _iInputNumberOfEvents,      // Number of events in input buffer
		char               * _pEventWindowBuffer,        // Event window buffer of this stream
		int                  _iWindowLength,             // Length of current events window
		int                  _iRemainingCount,           // Remaining free slots in Window buffer
		GpuKernelMetaEvent * _pOtherStreamMetaEvent,     // Meta event for other stream
		char               * _pOtherEventWindowBuffer,   // Event window buffer of other stream
		int                  _iOtherWindowLength,        // Length of current events window of other stream
		int                  _iOtherRemainingCount,      // Remaining free slots in Window buffer of other stream
		GpuKernelFilter    * _pOnCompareFilter,          // OnCompare filter buffer - pre-copied at initialization
		uint64_t             _iWithInTime,               // WithIn time in milliseconds
		GpuKernelMetaEvent * _pOutputStreamMetaEvent,    // Meta event for output stream
		char               * _pResultsBuffer,            // Resulting events buffer for this stream
		AttributeMappings  * _pOutputAttribMappings,     // Output event attribute mappings
		int                  _iEventsPerBlock            // number of events allocated per block
)
{
	// avoid out of bound threads
	if(threadIdx.x >= _iEventsPerBlock || threadIdx.y > 0 || blockIdx.y > 0)
		return;

	if((blockIdx.x == _iInputNumberOfEvents / _iEventsPerBlock) && // last thread block
			(threadIdx.x >= _iInputNumberOfEvents % _iEventsPerBlock)) // extra threads
	{
		return;
	}

	// get assigned event
	int iEventIdx = (blockIdx.x * _iEventsPerBlock) + threadIdx.x;

	// get in event starting position
	char * pInEventBuffer = _pInputEventBuffer + (_pInputMetaEvent->i_SizeOfEventInBytes * iEventIdx);

	// output to results buffer [in event, expired event]
	// {other stream event size * other window size} * 2 (for in/exp)
	int iOutputSegmentSize = _pOutputStreamMetaEvent->i_SizeOfEventInBytes * _iOtherWindowLength * 2;

	char * pResultsInEventBufferSegment = _pResultsBuffer + (iOutputSegmentSize * iEventIdx);
	char * pResultsExpiredEventBufferSegment = pResultsInEventBufferSegment + (iOutputSegmentSize / 2);

	char * pExpiredEventBuffer = NULL;
	GpuEvent * pExpiredEvent = NULL;

	GpuEvent * pInEvent = (GpuEvent*) pInEventBuffer;

	// calculate in/expired event pair for this event

	if(iEventIdx >= _iRemainingCount)
	{
		if(iEventIdx < _iWindowLength)
		{
			// in window buffer
			char * pExpiredOutEventInWindowBuffer = _pEventWindowBuffer + (_pInputMetaEvent->i_SizeOfEventInBytes * (iEventIdx - _iRemainingCount));

			GpuEvent * pWindowEvent = (GpuEvent*) pExpiredOutEventInWindowBuffer;
			if(pWindowEvent->i_Type != GpuEvent::NONE) // if window event is filled
			{
				pExpiredEventBuffer = pExpiredOutEventInWindowBuffer;

			}
			else
			{
				// no expiring event
			}
		}
		else
		{
			// in input event buffer
			char * pExpiredOutEventInInputBuffer = _pInputEventBuffer + (_pInputMetaEvent->i_SizeOfEventInBytes * (iEventIdx - _iWindowLength));
			pExpiredEventBuffer = pExpiredOutEventInInputBuffer;
		}
	}
	else
	{
		// [NULL,inEvent]
		// no expiring event
	}

	// get all matching event for in event from other window buffer and copy them to output event buffer

	// for each events in other window
	int iOtherWindowFillCount  = _iOtherWindowLength - _iOtherRemainingCount;
	int iMatchedCount = 0;
	for(int i=0; i<iOtherWindowFillCount; ++i)
	{
		// get other window event
		char * pOtherWindowEventBuffer = _pOtherEventWindowBuffer + (_pOtherStreamMetaEvent->i_SizeOfEventInBytes * i);
		GpuEvent * pOtherWindowEvent = (GpuEvent*) pOtherWindowEventBuffer;

		// get buffer position for in event matching results
		char * pResultInMatchingEventBuffer = pResultsInEventBufferSegment + (_pOutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
		GpuEvent * pResultInMatchingEvent = (GpuEvent*) pResultInMatchingEventBuffer;

		if(pInEvent->i_Sequence > pOtherWindowEvent->i_Sequence &&
				(pInEvent->i_Timestamp - pOtherWindowEvent->i_Timestamp) <= _iWithInTime)
		{
			ExpressionEvalParameters mExpressionParam;
			mExpressionParam.p_OnCompare = _pOnCompareFilter;
			mExpressionParam.a_Meta[0] = _pOtherStreamMetaEvent;
			mExpressionParam.a_Event[0] = pOtherWindowEventBuffer;
			mExpressionParam.a_Meta[1] = _pInputMetaEvent;
			mExpressionParam.a_Event[1] = pInEventBuffer;
			mExpressionParam.i_CurrentIndex = 0;

			bool bOnCompareMatched = Evaluate(mExpressionParam);
			if(bOnCompareMatched)
			{
				// copy output event to buffer - map attributes from input streams to output stream
				pResultInMatchingEvent->i_Type = GpuEvent::CURRENT;
				pResultInMatchingEvent->i_Sequence = pInEvent->i_Sequence;
				pResultInMatchingEvent->i_Timestamp = pInEvent->i_Timestamp;

				for(int m=0; m < _pOutputAttribMappings->i_MappingCount; ++m)
				{
					int iFromStreamIndex = _pOutputAttribMappings->p_Mappings[m].from[AttributeMapping::STREAM_INDEX];
					int iFromAttrib = _pOutputAttribMappings->p_Mappings[m].from[AttributeMapping::ATTRIBUTE_INDEX];
					int iTo = _pOutputAttribMappings->p_Mappings[m].to;

					memcpy(
						pResultInMatchingEventBuffer + _pOutputStreamMetaEvent->p_Attributes[iTo].i_Position, // to
						mExpressionParam.a_Event[iFromStreamIndex] + mExpressionParam.a_Meta[iFromStreamIndex]->p_Attributes[iFromAttrib].i_Position, // from
						mExpressionParam.a_Meta[iFromStreamIndex]->p_Attributes[iFromAttrib].i_Length // size
					);
				}

				iMatchedCount++;
			}
		}
		else
		{
			// cannot continue, last result event for this segment
			pResultInMatchingEvent->i_Type = GpuEvent::RESET;
			break;
		}

		if(iMatchedCount < iOtherWindowFillCount || iOtherWindowFillCount == 0)
		{
			char * pResultInMatchingEventBuffer = pResultsInEventBufferSegment + (_pOutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
			GpuEvent * pResultInMatchingEvent = (GpuEvent*) pResultInMatchingEventBuffer;
			pResultInMatchingEvent->i_Type = GpuEvent::RESET;
		}
	}

	if(pExpiredEventBuffer != NULL)
	{
		pExpiredEvent = (GpuEvent*) pExpiredEventBuffer;

		iMatchedCount = 0;
		// for each events in other window
		for(int i=0; i<iOtherWindowFillCount; ++i)
		{
			// get other window event
			char * pOtherWindowEventBuffer = _pOtherEventWindowBuffer + (_pOtherStreamMetaEvent->i_SizeOfEventInBytes * i);
			GpuEvent * pOtherWindowEvent = (GpuEvent*) pOtherWindowEventBuffer;

			// get buffer position for expire event matching results
			char * pResultExpireMatchingEventBuffer = pResultsExpiredEventBufferSegment + (_pOutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
			GpuEvent * pResultExpireMatchingEvent = (GpuEvent*) pResultExpireMatchingEventBuffer;

			if(pExpiredEvent->i_Sequence < pOtherWindowEvent->i_Sequence &&
					(pOtherWindowEvent->i_Timestamp - pExpiredEvent->i_Timestamp) <= _iWithInTime)
			{
				ExpressionEvalParameters mExpressionParam;
				mExpressionParam.p_OnCompare = _pOnCompareFilter;
				mExpressionParam.a_Meta[0] = _pOtherStreamMetaEvent;
				mExpressionParam.a_Event[0] = pOtherWindowEventBuffer;
				mExpressionParam.a_Meta[1] = _pInputMetaEvent;
				mExpressionParam.a_Event[1] = pExpiredEventBuffer;
				mExpressionParam.i_CurrentIndex = 0;

				bool bOnCompareMatched = Evaluate(mExpressionParam);
				if(bOnCompareMatched)
				{
					// copy output event to buffer - map attributes from input streams to output stream
					pResultExpireMatchingEvent->i_Type = GpuEvent::EXPIRED;
					pResultExpireMatchingEvent->i_Sequence = pExpiredEvent->i_Sequence;
					pResultExpireMatchingEvent->i_Timestamp = pExpiredEvent->i_Timestamp;

					for(int m=0; m < _pOutputAttribMappings->i_MappingCount; ++m)
					{
						int iFromStreamIndex = _pOutputAttribMappings->p_Mappings[m].from[AttributeMapping::STREAM_INDEX];
						int iFromAttrib = _pOutputAttribMappings->p_Mappings[m].from[AttributeMapping::ATTRIBUTE_INDEX];
						int iTo = _pOutputAttribMappings->p_Mappings[m].to;

						memcpy(
								pResultExpireMatchingEventBuffer + _pOutputStreamMetaEvent->p_Attributes[iTo].i_Position, // to
								mExpressionParam.a_Event[iFromStreamIndex] + mExpressionParam.a_Meta[iFromStreamIndex]->p_Attributes[iFromAttrib].i_Position, // from
								mExpressionParam.a_Meta[iFromStreamIndex]->p_Attributes[iFromAttrib].i_Length // size
						);
					}

					iMatchedCount++;
				}
			}
			else
			{
				// cannot continue, last result event for this segment
				pResultExpireMatchingEvent->i_Type = GpuEvent::RESET;
				break;
			}
		}

		if(iMatchedCount < iOtherWindowFillCount || iOtherWindowFillCount == 0)
		{
			char * pResultExpireMatchingEventBuffer = pResultsExpiredEventBufferSegment + (_pOutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
			GpuEvent * pResultExpireMatchingEvent = (GpuEvent*) pResultExpireMatchingEventBuffer;
			pResultExpireMatchingEvent->i_Type = GpuEvent::RESET;
		}
	}

}

//__global__
//void ProcessEventsJoinRightTriggerCurrentOn(
//		char               * _pInputEventBuffer,         // input events buffer
//		GpuKernelMetaEvent * _pInputMetaEvent,           // Meta event for input events
//		int                  _iInputNumberOfEvents,      // Number of events in input buffer
//		char               * _pEventWindowBuffer,        // Event window buffer of this stream
//		int                  _iWindowLength,             // Length of current events window
//		int                  _iRemainingCount,           // Remaining free slots in Window buffer
//		GpuKernelMetaEvent * _pOtherStreamMetaEvent,     // Meta event for other stream
//		char               * _pOtherEventWindowBuffer,   // Event window buffer of other stream
//		int                  _iOtherWindowLength,        // Length of current events window of other stream
//		int                  _iOtherRemainingCount,      // Remaining free slots in Window buffer of other stream
//		GpuKernelFilter    * _pOnCompareFilter,          // OnCompare filter buffer - pre-copied at initialization
//		uint64_t             _iWithInTime,               // WithIn time in milliseconds
//		GpuKernelMetaEvent * _pOutputStreamMetaEvent,    // Meta event for output stream
//		char               * _pResultsBuffer,            // Resulting events buffer for this stream
//		AttributeMappings  * _pOutputAttribMappings,     // Output event attribute mappings
//		int                  _iEventsPerBlock,           // number of events allocated per block
//		int                  _iWorkSize                  // Number of events in window process by this kernel
//)

__global__
void
__launch_bounds__(MY_KERNEL_MAX_THREADS, MY_KERNEL_MIN_BLOCKS)
ProcessEventsJoinRightTriggerCurrentOn(
		JoinKernelParameters * _pParameters,
		int                   _iInputNumberOfEvents,      // Number of events in input buffer
		int                   _iRemainingCount,           // Remaining free slots in Window buffer
		int                   _iOtherRemainingCount      // Remaining free slots in Window buffer of other stream
)
{
	// avoid out of bound threads
	if(threadIdx.x >= _pParameters->i_EventsPerBlock || threadIdx.y > 0 || blockIdx.y > 0)
		return;

	int iWorkerCount = ceil((float)_pParameters->i_OtherWindowLength / _pParameters->i_WorkSize);

	if((blockIdx.x == (_iInputNumberOfEvents * iWorkerCount) / _pParameters->i_EventsPerBlock) && // last thread block
			(threadIdx.x >= (_iInputNumberOfEvents * iWorkerCount) % _pParameters->i_EventsPerBlock)) // extra threads
	{
		return;
	}

	extern __shared__ char p_SharedInputEventBuffer[];

	// get assigned event
	int iGlobalThreadIdx = (blockIdx.x * _pParameters->i_EventsPerBlock) + threadIdx.x;

	// get in buffer index
	int iInEventIndex = iGlobalThreadIdx / iWorkerCount;
	int iWindowStartEventIndex = (iGlobalThreadIdx % iWorkerCount) * _pParameters->i_WorkSize;

	// get in event starting position
//	char * pInEventBuffer = _pParameters->p_InputEventBuffer + (_pParameters->p_InputMetaEvent->i_SizeOfEventInBytes * iInEventIndex);
	char * pSharedInEventBuffer = p_SharedInputEventBuffer + (_pParameters->p_InputMetaEvent->i_SizeOfEventInBytes * (threadIdx.x / iWorkerCount));
	if(threadIdx.x % iWorkerCount == 0)
	{
		memcpy(pSharedInEventBuffer,
			_pParameters->p_InputEventBuffer + (_pParameters->p_InputMetaEvent->i_SizeOfEventInBytes * iInEventIndex),
			_pParameters->p_InputMetaEvent->i_SizeOfEventInBytes);
	}
	__syncthreads();

	// output to results buffer [in event, expired event]
	// {other stream event size * other window size} * 2 (for in/exp)
//	int iOutputSegmentSizePerEvent = _pOutputStreamMetaEvent->i_SizeOfEventInBytes * _iOtherWindowLength;

	char * pResultsInEventBufferSegment = _pParameters->p_ResultsBuffer + (iGlobalThreadIdx * _pParameters->i_WorkSize * _pParameters->p_OutputStreamMetaEvent->i_SizeOfEventInBytes);
//			+ (iWindowStartEventIndex * _pOutputStreamMetaEvent->i_SizeOfEventInBytes);

	GpuEvent * pInEvent = (GpuEvent*) pSharedInEventBuffer;

//	memset(pResultsInEventBufferSegment, 0, _pParameters->i_WorkSize * _pParameters->p_OutputStreamMetaEvent->i_SizeOfEventInBytes);

	// get all matching event for in event from other window buffer and copy them to output event buffer

	// for each events in other window
	int iOtherWindowFillCount  = _pParameters->i_OtherWindowLength - _iOtherRemainingCount;

	if(iWindowStartEventIndex < iOtherWindowFillCount)
	{
		int iWindowEndEventIndex = min(iWindowStartEventIndex + _pParameters->i_WorkSize, iOtherWindowFillCount);

		int iMatchedCount = 0;

		// get buffer position for in event matching results
		char * pResultInMatchingEventBuffer = pResultsInEventBufferSegment + (_pParameters->p_OutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
		GpuEvent * pResultInMatchingEvent = (GpuEvent*) pResultInMatchingEventBuffer;

		for(int i=iWindowStartEventIndex; i<iWindowEndEventIndex; ++i)
		{
			// get other window event
			char * pOtherWindowEventBuffer = _pParameters->p_OtherEventWindowBuffer + (_pParameters->p_OtherStreamMetaEvent->i_SizeOfEventInBytes * i);
			GpuEvent * pOtherWindowEvent = (GpuEvent*) pOtherWindowEventBuffer;

			if(pInEvent->i_Sequence > pOtherWindowEvent->i_Sequence &&
					(pInEvent->i_Timestamp - pOtherWindowEvent->i_Timestamp) <= _pParameters->i_WithInTime)
			{

				ExpressionEvalParameters mExpressionParam;
				mExpressionParam.p_OnCompare = _pParameters->p_OnCompareFilter;
				mExpressionParam.a_Meta[0] = _pParameters->p_OtherStreamMetaEvent;
				mExpressionParam.a_Event[0] = pOtherWindowEventBuffer;
				mExpressionParam.a_Meta[1] = _pParameters->p_InputMetaEvent;
				mExpressionParam.a_Event[1] = pSharedInEventBuffer;
				mExpressionParam.i_CurrentIndex = 0;

				bool bOnCompareMatched = Evaluate(mExpressionParam);
				if(bOnCompareMatched)
				{
					// copy output event to buffer - map attributes from input streams to output stream
					pResultInMatchingEvent->i_Type = GpuEvent::CURRENT;
					pResultInMatchingEvent->i_Sequence = pInEvent->i_Sequence;
					pResultInMatchingEvent->i_Timestamp = pInEvent->i_Timestamp;

					#pragma __unroll__
					for(int m=0; m < _pParameters->p_OutputAttribMappings->i_MappingCount; ++m)
					{
						int iFromStreamIndex = _pParameters->p_OutputAttribMappings->p_Mappings[m].from[AttributeMapping::STREAM_INDEX];
						int iFromAttrib = _pParameters->p_OutputAttribMappings->p_Mappings[m].from[AttributeMapping::ATTRIBUTE_INDEX];
						int iTo = _pParameters->p_OutputAttribMappings->p_Mappings[m].to;

						memcpy(
								pResultInMatchingEventBuffer + _pParameters->p_OutputStreamMetaEvent->p_Attributes[iTo].i_Position, // to
								mExpressionParam.a_Event[iFromStreamIndex] + mExpressionParam.a_Meta[iFromStreamIndex]->p_Attributes[iFromAttrib].i_Position, // from
								mExpressionParam.a_Meta[iFromStreamIndex]->p_Attributes[iFromAttrib].i_Length // size
						);
					}

					iMatchedCount++;

					pResultInMatchingEventBuffer = pResultsInEventBufferSegment + (_pParameters->p_OutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
					pResultInMatchingEvent = (GpuEvent*) pResultInMatchingEventBuffer;
				}
			}
			else
			{
				pResultInMatchingEventBuffer = pResultsInEventBufferSegment + (_pParameters->p_OutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
				pResultInMatchingEvent = (GpuEvent*) pResultInMatchingEventBuffer;

				// cannot continue, last result event for this segment
				pResultInMatchingEvent->i_Type = GpuEvent::RESET;
				break;
			}
		}

		if(iMatchedCount < (iWindowEndEventIndex - iWindowStartEventIndex))
		{
			char * pResultInMatchingEventBuffer = pResultsInEventBufferSegment + (_pParameters->p_OutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
			GpuEvent * pResultInMatchingEvent = (GpuEvent*) pResultInMatchingEventBuffer;
			pResultInMatchingEvent->i_Type = GpuEvent::RESET;
		}
	}
	else
	{
		GpuEvent * pResultInMatchingEvent = (GpuEvent*) pResultsInEventBufferSegment;
		pResultInMatchingEvent->i_Type = GpuEvent::RESET;
	}

}

__global__
void
__launch_bounds__(MY_KERNEL_MAX_THREADS, MY_KERNEL_MIN_BLOCKS)
ProcessEventsJoinRightTriggerExpireOn(
		char               * _pInputEventBuffer,         // input events buffer
		GpuKernelMetaEvent * _pInputMetaEvent,           // Meta event for input events
		int                  _iInputNumberOfEvents,      // Number of events in input buffer
		char               * _pEventWindowBuffer,        // Event window buffer of this stream
		int                  _iWindowLength,             // Length of current events window
		int                  _iRemainingCount,           // Remaining free slots in Window buffer
		GpuKernelMetaEvent * _pOtherStreamMetaEvent,     // Meta event for other stream
		char               * _pOtherEventWindowBuffer,   // Event window buffer of other stream
		int                  _iOtherWindowLength,        // Length of current events window of other stream
		int                  _iOtherRemainingCount,      // Remaining free slots in Window buffer of other stream
		GpuKernelFilter    * _pOnCompareFilter,          // OnCompare filter buffer - pre-copied at initialization
		uint64_t             _iWithInTime,               // WithIn time in milliseconds
		GpuKernelMetaEvent * _pOutputStreamMetaEvent,    // Meta event for output stream
		char               * _pResultsBuffer,            // Resulting events buffer for this stream
		AttributeMappings  * _pOutputAttribMappings,     // Output event attribute mappings
		int                  _iEventsPerBlock            // number of events allocated per block
)
{
	// avoid out of bound threads
	if(threadIdx.x >= _iEventsPerBlock || threadIdx.y > 0 || blockIdx.y > 0)
		return;

	if((blockIdx.x == _iInputNumberOfEvents / _iEventsPerBlock) && // last thread block
			(threadIdx.x >= _iInputNumberOfEvents % _iEventsPerBlock)) // extra threads
	{
		return;
	}

	// get assigned event
	int iEventIdx = (blockIdx.x * _iEventsPerBlock) + threadIdx.x;

	// output to results buffer [in event, expired event]
	// {other stream event size * other window size} * 2 (for in/exp)
	int iOutputSegmentSize = _pOutputStreamMetaEvent->i_SizeOfEventInBytes * _iOtherWindowLength;

	char * pResultsExpiredEventBufferSegment = _pResultsBuffer + (iOutputSegmentSize * iEventIdx);

	char * pExpiredEventBuffer = NULL;
	GpuEvent * pExpiredEvent = NULL;

	// calculate in/expired event pair for this event

	if(iEventIdx >= _iRemainingCount)
	{
		if(iEventIdx < _iWindowLength)
		{
			// in window buffer
			char * pExpiredOutEventInWindowBuffer = _pEventWindowBuffer + (_pInputMetaEvent->i_SizeOfEventInBytes * (iEventIdx - _iRemainingCount));

			GpuEvent * pWindowEvent = (GpuEvent*) pExpiredOutEventInWindowBuffer;
			if(pWindowEvent->i_Type != GpuEvent::NONE) // if window event is filled
			{
				pExpiredEventBuffer = pExpiredOutEventInWindowBuffer;

			}
			else
			{
				// no expiring event
			}
		}
		else
		{
			// in input event buffer
			char * pExpiredOutEventInInputBuffer = _pInputEventBuffer + (_pInputMetaEvent->i_SizeOfEventInBytes * (iEventIdx - _iWindowLength));
			pExpiredEventBuffer = pExpiredOutEventInInputBuffer;
		}
	}
	else
	{
		// [NULL,inEvent]
		// no expiring event
	}

	if(pExpiredEventBuffer != NULL)
	{

		// get all matching event for in event from other window buffer and copy them to output event buffer

		pExpiredEvent = (GpuEvent*) pExpiredEventBuffer;

		// for each events in other window
		int iOtherWindowFillCount  = _iOtherWindowLength - _iOtherRemainingCount;
		int iMatchedCount = 0;

		// for each events in other window
		for(int i=0; i<iOtherWindowFillCount; ++i)
		{
			// get other window event
			char * pOtherWindowEventBuffer = _pOtherEventWindowBuffer + (_pOtherStreamMetaEvent->i_SizeOfEventInBytes * i);
			GpuEvent * pOtherWindowEvent = (GpuEvent*) pOtherWindowEventBuffer;

			// get buffer position for expire event matching results
			char * pResultExpireMatchingEventBuffer = pResultsExpiredEventBufferSegment + (_pOutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
			GpuEvent * pResultExpireMatchingEvent = (GpuEvent*) pResultExpireMatchingEventBuffer;

			if(pExpiredEvent->i_Sequence < pOtherWindowEvent->i_Sequence &&
					(pOtherWindowEvent->i_Timestamp - pExpiredEvent->i_Timestamp) <= _iWithInTime)
			{
				ExpressionEvalParameters mExpressionParam;
				mExpressionParam.p_OnCompare = _pOnCompareFilter;
				mExpressionParam.a_Meta[0] = _pOtherStreamMetaEvent;
				mExpressionParam.a_Event[0] = pOtherWindowEventBuffer;
				mExpressionParam.a_Meta[1] = _pInputMetaEvent;
				mExpressionParam.a_Event[1] = pExpiredEventBuffer;
				mExpressionParam.i_CurrentIndex = 0;

				bool bOnCompareMatched = Evaluate(mExpressionParam);
				if(bOnCompareMatched)
				{
					// copy output event to buffer - map attributes from input streams to output stream
					pResultExpireMatchingEvent->i_Type = GpuEvent::EXPIRED;
					pResultExpireMatchingEvent->i_Sequence = pExpiredEvent->i_Sequence;
					pResultExpireMatchingEvent->i_Timestamp = pExpiredEvent->i_Timestamp;

					for(int m=0; m < _pOutputAttribMappings->i_MappingCount; ++m)
					{
						int iFromStreamIndex = _pOutputAttribMappings->p_Mappings[m].from[AttributeMapping::STREAM_INDEX];
						int iFromAttrib = _pOutputAttribMappings->p_Mappings[m].from[AttributeMapping::ATTRIBUTE_INDEX];
						int iTo = _pOutputAttribMappings->p_Mappings[m].to;

						memcpy(
								pResultExpireMatchingEventBuffer + _pOutputStreamMetaEvent->p_Attributes[iTo].i_Position, // to
								mExpressionParam.a_Event[iFromStreamIndex] + mExpressionParam.a_Meta[iFromStreamIndex]->p_Attributes[iFromAttrib].i_Position, // from
								mExpressionParam.a_Meta[iFromStreamIndex]->p_Attributes[iFromAttrib].i_Length // size
						);
					}

					iMatchedCount++;
				}
			}
			else
			{
				// cannot continue, last result event for this segment
				pResultExpireMatchingEvent->i_Type = GpuEvent::RESET;
				break;
			}
		}

		if(iMatchedCount < iOtherWindowFillCount || iOtherWindowFillCount == 0)
		{
			char * pResultExpireMatchingEventBuffer = pResultsExpiredEventBufferSegment + (_pOutputStreamMetaEvent->i_SizeOfEventInBytes * iMatchedCount);
			GpuEvent * pResultExpireMatchingEvent = (GpuEvent*) pResultExpireMatchingEventBuffer;
			pResultExpireMatchingEvent->i_Type = GpuEvent::RESET;
		}
	}

}

__global__
void
__launch_bounds__(MY_KERNEL_MAX_THREADS, MY_KERNEL_MIN_BLOCKS)
JoinSetWindowState(
		char               * _pInputEventBuffer,     // original input events buffer
		int                  _iNumberOfEvents,       // Number of events in input buffer (matched + not matched)
		char               * _pEventWindowBuffer,    // Event window buffer
		int                  _iWindowLength,         // Length of current events window
		int                  _iRemainingCount,       // Remaining free slots in Window buffer
		int                  _iMaxEventCount,        // used for setting results array
		int                  _iSizeOfEvent,          // Size of an event
		int                  _iEventsPerBlock        // number of events allocated per block
)
{
	// avoid out of bound threads
	if(threadIdx.x >= _iEventsPerBlock || threadIdx.y > 0 || blockIdx.y > 0)
		return;

	if((blockIdx.x == _iNumberOfEvents / _iEventsPerBlock) && // last thread block
			(threadIdx.x >= _iNumberOfEvents % _iEventsPerBlock)) // extra threads
	{
		return;
	}

	// get assigned event
	int iEventIdx = (blockIdx.x * _iEventsPerBlock) + threadIdx.x;

	// get in event starting position
	char * pInEventBuffer = _pInputEventBuffer + (_iSizeOfEvent * iEventIdx);

	if(_iNumberOfEvents < _iWindowLength)
	{
		int iWindowPositionShift = _iWindowLength - _iNumberOfEvents;

		if(_iRemainingCount < _iNumberOfEvents)
		{
			int iExitEventCount = _iNumberOfEvents - _iRemainingCount;

			// calculate start and end window buffer positions
			int iStart = iEventIdx + iWindowPositionShift;
			int iEnd = iStart;
			int iPrevToEnd = iEnd;
			while(iEnd >= 0)
			{
				char * pDestinationEventBuffer = _pEventWindowBuffer + (_iSizeOfEvent * iEnd);
				GpuEvent * pDestinationEvent = (GpuEvent*) pDestinationEventBuffer;

				if(pDestinationEvent->i_Type != GpuEvent::NONE) // there is an event in destination position
				{
					iPrevToEnd = iEnd;
					iEnd -= iExitEventCount;
				}
				else
				{
					break;
				}

			}

			iEnd = (iEnd < 0 ? iPrevToEnd : iEnd);

			// work back from end while copying events
			while(iEnd < iStart)
			{
				char * pDestinationEventBuffer = _pEventWindowBuffer + (_iSizeOfEvent * iEnd);
				GpuEvent * pDestinationEvent = (GpuEvent*) pDestinationEventBuffer;

				char * pSourceEventBuffer = _pEventWindowBuffer + (_iSizeOfEvent * (iEnd + iExitEventCount));

				memcpy(pDestinationEventBuffer, pSourceEventBuffer, _iSizeOfEvent);
				pDestinationEvent->i_Type = GpuEvent::EXPIRED;

				iEnd += iExitEventCount;
			}

			// iEnd == iStart
			if(iStart >= 0)
			{
				char * pDestinationEventBuffer = _pEventWindowBuffer + (_iSizeOfEvent * iStart);
				GpuEvent * pDestinationEvent = (GpuEvent*) pDestinationEventBuffer;
				memcpy(pDestinationEventBuffer, pInEventBuffer, _iSizeOfEvent);
				pDestinationEvent->i_Type = GpuEvent::EXPIRED;
			}
		}
		else
		{
			// just copy event to window
			iWindowPositionShift -= (_iRemainingCount - _iNumberOfEvents);

			char * pWindowEventBuffer = _pEventWindowBuffer + (_iSizeOfEvent * (iEventIdx + iWindowPositionShift));

			memcpy(pWindowEventBuffer, pInEventBuffer, _iSizeOfEvent);
			GpuEvent * pExpiredEvent = (GpuEvent*) pWindowEventBuffer;
			pExpiredEvent->i_Type = GpuEvent::EXPIRED;
		}
	}
	else
	{
		int iWindowPositionShift = _iNumberOfEvents - _iWindowLength;

		if(iEventIdx >= iWindowPositionShift)
		{
			char * pWindowEventBuffer = _pEventWindowBuffer + (_iSizeOfEvent * (iEventIdx - iWindowPositionShift));

			memcpy(pWindowEventBuffer, pInEventBuffer, _iSizeOfEvent);
			GpuEvent * pExpiredEvent = (GpuEvent*) pWindowEventBuffer;
			pExpiredEvent->i_Type = GpuEvent::EXPIRED;
		}
	}
}


// ======================================================================================================================

GpuJoinKernel::GpuJoinKernel(GpuProcessor * _pProc, GpuProcessorContext * _pLeftContext, GpuProcessorContext * _pRightContext,
		int _iThreadBlockSize, int _iLeftWindowSize, int _iRightWindowSize, FILE * _fpLeftLog, FILE * _fpRightLog) :
	GpuKernel(_pProc, _pLeftContext->GetDeviceId(), _iThreadBlockSize, _fpLeftLog),
	p_LeftContext(_pLeftContext),
	p_RightContext(_pRightContext),
	i_LeftInputBufferIndex(0),
	i_RightInputBufferIndex(0),
	p_LeftInputEventBuffer(NULL),
	p_RightInputEventBuffer(NULL),
	p_LeftWindowEventBuffer(NULL),
	p_RightWindowEventBuffer(NULL),
	p_LeftResultEventBuffer(NULL),
	p_RightResultEventBuffer(NULL),
	p_DeviceOnCompareFilter(NULL),
	p_DeviceParametersLeft(NULL),
	p_DeviceParametersRight(NULL),
	i_LeftStreamWindowSize(_iLeftWindowSize),
	i_RightStreamWindowSize(_iRightWindowSize),
//	i_LeftRemainingCount(_iLeftWindowSize),
//	i_RightRemainingCount(_iRightWindowSize),
	i_LeftNumEventPerSegment(0),
	i_RightNumEventPerSegment(0),
	b_LeftFirstKernel(true),
	b_RightFirstKernel(true),
	b_LeftDeviceSet(false),
	b_RightDeviceSet(false),
	i_LeftThreadWorkSize(_iRightWindowSize),
	i_RightThreadWorkSize(_iLeftWindowSize),
	i_LeftThreadWorkerCount(0),
	i_RightThreadWorkerCount(0),
	i_InitializedStreamCount(0),
	fp_LeftLog(_fpLeftLog),
	fp_RightLog(_fpRightLog)
{
	p_JoinProcessor = (GpuJoinProcessor*) _pProc;
	pthread_mutex_init(&mtx_Lock, NULL);
}

GpuJoinKernel::~GpuJoinKernel()
{
	fprintf(fp_LeftLog, "[GpuJoinKernel] destroy\n");
	fflush(fp_LeftLog);
	fprintf(fp_RightLog, "[GpuJoinKernel] destroy\n");
	fflush(fp_RightLog);

	CUDA_CHECK_RETURN(hipFree(p_DeviceOnCompareFilter));
	p_DeviceOnCompareFilter = NULL;

	if(p_DeviceOutputAttributeMapping)
	{
		CUDA_CHECK_RETURN(hipFree(p_DeviceOutputAttributeMapping));
		p_DeviceOutputAttributeMapping = NULL;
	}

	if(p_DeviceParametersLeft)
	{
		CUDA_CHECK_RETURN(hipFree(p_DeviceParametersLeft));
		p_DeviceParametersLeft = NULL;
	}

	if(p_DeviceParametersRight)
	{
		CUDA_CHECK_RETURN(hipFree(p_DeviceParametersRight));
		p_DeviceParametersRight = NULL;
	}

	pthread_mutex_destroy(&mtx_Lock);
}

bool GpuJoinKernel::Initialize(int _iStreamIndex, GpuMetaEvent * _pMetaEvent, int _iInputEventBufferSize)
{
	if(_iStreamIndex == 0)
	{
		fprintf(fp_LeftLog, "[GpuJoinKernel] Initialize : StreamIndex=%d LeftTrigger=%d RightTrigger=%d CurrentOn=%d ExpireOn=%d\n",
				_iStreamIndex, p_JoinProcessor->GetLeftTrigger(), p_JoinProcessor->GetRightTrigger(),
				p_JoinProcessor->GetCurrentOn(), p_JoinProcessor->GetExpiredOn());
		fflush(fp_LeftLog);

		// set input event buffer
		fprintf(fp_LeftLog, "[GpuJoinKernel] Left InpuEventBufferIndex=%d\n", i_LeftInputBufferIndex);
		fflush(fp_LeftLog);
		p_LeftInputEventBuffer = (GpuStreamEventBuffer*) p_LeftContext->GetEventBuffer(i_LeftInputBufferIndex);
		p_LeftInputEventBuffer->Print();

		// left event window

		p_LeftWindowEventBuffer = new GpuWindowEventBuffer("LeftWindowEventBuffer", p_LeftContext->GetDeviceId(), _pMetaEvent, fp_LeftLog);
		p_LeftWindowEventBuffer->CreateEventBuffer(i_LeftStreamWindowSize);

		fprintf(fp_LeftLog, "[GpuJoinKernel] Created device left window buffer : Length=%d Size=%d bytes\n", i_LeftStreamWindowSize,
				p_LeftWindowEventBuffer->GetEventBufferSizeInBytes());
		fflush(fp_LeftLog);

		fprintf(fp_LeftLog, "[GpuJoinKernel] initialize left window buffer data \n");
		fflush(fp_LeftLog);
		p_LeftWindowEventBuffer->Print();

		p_LeftWindowEventBuffer->ResetHostEventBuffer(0);

		char * pLeftHostWindowBuffer = p_LeftWindowEventBuffer->GetHostEventBuffer();
		char * pCurrentEvent;
		for(int i=0; i<i_LeftStreamWindowSize; ++i)
		{
			pCurrentEvent = pLeftHostWindowBuffer + (_pMetaEvent->i_SizeOfEventInBytes * i);
			GpuEvent * pGpuEvent = (GpuEvent*) pCurrentEvent;
			pGpuEvent->i_Type = GpuEvent::NONE;
		}
		p_LeftWindowEventBuffer->CopyToDevice(false);
		p_LeftWindowEventBuffer->Sync(0, false);

		i_InitializedStreamCount++;

		GpuUtils::PrintThreadInfo("GpuJoinKernel", fp_LeftLog);
	}
	else if(_iStreamIndex == 1)
	{
		fprintf(fp_RightLog, "[GpuJoinKernel] Initialize : StreamIndex=%d LeftTrigger=%d RightTrigger=%d CurrentOn=%d ExpireOn=%d\n",
				_iStreamIndex, p_JoinProcessor->GetLeftTrigger(), p_JoinProcessor->GetRightTrigger(),
				p_JoinProcessor->GetCurrentOn(), p_JoinProcessor->GetExpiredOn());
		fflush(fp_RightLog);

		fprintf(fp_RightLog, "[GpuJoinKernel] Right InpuEventBufferIndex=%d\n", i_RightInputBufferIndex);
		fflush(fp_RightLog);
		p_RightInputEventBuffer = (GpuStreamEventBuffer*) p_RightContext->GetEventBuffer(i_RightInputBufferIndex);
		p_RightInputEventBuffer->Print();

		// right event window

		p_RightWindowEventBuffer = new GpuWindowEventBuffer("RightWindowEventBuffer", p_RightContext->GetDeviceId(), _pMetaEvent, fp_RightLog);
		p_RightWindowEventBuffer->CreateEventBuffer(i_RightStreamWindowSize);

		fprintf(fp_RightLog, "[GpuJoinKernel] Created device right window buffer : Length=%d Size=%d bytes\n", i_RightStreamWindowSize,
				p_RightWindowEventBuffer->GetEventBufferSizeInBytes());
		fflush(fp_RightLog);

		fprintf(fp_RightLog, "[GpuJoinKernel] initialize right window buffer data \n");
		fflush(fp_RightLog);
		p_RightWindowEventBuffer->Print();

		p_RightWindowEventBuffer->ResetHostEventBuffer(0);

		char * pRightHostWindowBuffer = p_RightWindowEventBuffer->GetHostEventBuffer();
		char * pCurrentEvent;
		for(int i=0; i<i_RightStreamWindowSize; ++i)
		{
			pCurrentEvent = pRightHostWindowBuffer + (_pMetaEvent->i_SizeOfEventInBytes * i);
			GpuEvent * pGpuEvent = (GpuEvent*) pCurrentEvent;
			pGpuEvent->i_Type = GpuEvent::NONE;
		}
		p_RightWindowEventBuffer->CopyToDevice(false);
		p_RightWindowEventBuffer->Sync(0, false);

		i_InitializedStreamCount++;

		GpuUtils::PrintThreadInfo("GpuJoinKernel", fp_RightLog);
	}

	if(i_InitializedStreamCount == 2)
	{
		fprintf(fp_LeftLog, "[GpuJoinKernel] StreamId=%d Creating result event buffer\n", _iStreamIndex);
		fflush(fp_LeftLog);
		fprintf(fp_RightLog, "[GpuJoinKernel] StreamId=%d Creating result event buffer\n", _iStreamIndex);
		fflush(fp_RightLog);

		p_LeftResultEventBuffer = new GpuStreamEventBuffer("JoinLeftResultEventBuffer", p_LeftContext->GetDeviceId(), p_OutputStreamMeta, fp_LeftLog);
		if(p_JoinProcessor->GetLeftTrigger())
		{
			int iEventCount = 0;
			if(p_JoinProcessor->GetCurrentOn())
			{
				iEventCount += i_RightStreamWindowSize * p_LeftInputEventBuffer->GetMaxEventCount();
				i_LeftNumEventPerSegment = i_RightStreamWindowSize;
			}
			if(p_JoinProcessor->GetExpiredOn())
			{
				iEventCount += i_RightStreamWindowSize * p_LeftInputEventBuffer->GetMaxEventCount();
				i_LeftNumEventPerSegment += i_RightStreamWindowSize;
			}
			p_LeftResultEventBuffer->CreateEventBuffer(iEventCount);
			fprintf(fp_LeftLog, "[GpuJoinKernel] LeftResultEventBuffer created : Size=%d bytes\n", p_LeftResultEventBuffer->GetEventBufferSizeInBytes());
			fflush(fp_LeftLog);
		}
		p_LeftResultEventBuffer->Print();


		p_RightResultEventBuffer = new GpuStreamEventBuffer("JoinRightResultEventBuffer", p_RightContext->GetDeviceId(), p_OutputStreamMeta, fp_RightLog);
		if(p_JoinProcessor->GetRightTrigger())
		{
			int iEventCount = 0;
			if(p_JoinProcessor->GetCurrentOn())
			{
				iEventCount += i_LeftStreamWindowSize * p_RightInputEventBuffer->GetMaxEventCount();
				i_RightNumEventPerSegment = i_LeftStreamWindowSize;
			}
			if(p_JoinProcessor->GetExpiredOn())
			{
				iEventCount += i_LeftStreamWindowSize * p_RightInputEventBuffer->GetMaxEventCount();
				i_RightNumEventPerSegment += i_LeftStreamWindowSize;
			}

			p_RightResultEventBuffer->CreateEventBuffer(iEventCount);
			fprintf(fp_RightLog, "[GpuJoinKernel] RightResultEventBuffer created : Size=%d bytes\n", p_RightResultEventBuffer->GetEventBufferSizeInBytes());
			fflush(fp_RightLog);
		}
		p_RightResultEventBuffer->Print();


		fprintf(fp_LeftLog, "[GpuJoinKernel] Copying OnCompare filter to device \n");
		fflush(fp_LeftLog);
		fprintf(fp_RightLog, "[GpuJoinKernel] Copying OnCompare filter to device \n");
		fflush(fp_RightLog);

		CUDA_CHECK_RETURN(hipMalloc(
				(void**) &p_DeviceOnCompareFilter,
				sizeof(GpuKernelFilter)));

		GpuKernelFilter * apHostFilters = (GpuKernelFilter *) malloc(sizeof(GpuKernelFilter));

		apHostFilters->i_NodeCount = p_JoinProcessor->i_NodeCount;
		apHostFilters->ap_ExecutorNodes = NULL;

		CUDA_CHECK_RETURN(hipMalloc(
				(void**) &apHostFilters->ap_ExecutorNodes,
				sizeof(ExecutorNode) * p_JoinProcessor->i_NodeCount));

		CUDA_CHECK_RETURN(hipMemcpy(
				apHostFilters->ap_ExecutorNodes,
				p_JoinProcessor->ap_ExecutorNodes,
				sizeof(ExecutorNode) * p_JoinProcessor->i_NodeCount,
				hipMemcpyHostToDevice));

		CUDA_CHECK_RETURN(hipMemcpy(
				p_DeviceOnCompareFilter,
				apHostFilters,
				sizeof(GpuKernelFilter),
				hipMemcpyHostToDevice));

		CUDA_CHECK_RETURN(hipPeekAtLastError());
		CUDA_CHECK_RETURN(hipDeviceSynchronize());

		free(apHostFilters);
		apHostFilters = NULL;

		// copy Output mappings
		if(p_HostOutputAttributeMapping)
		{
			fprintf(fp_LeftLog, "[GpuJoinKernel] Copying AttributeMappings to device \n");
			fflush(fp_LeftLog);
			fprintf(fp_RightLog, "[GpuJoinKernel] Copying AttributeMappings to device \n");
			fflush(fp_RightLog);

			fprintf(fp_LeftLog, "[GpuJoinKernel] AttributeMapCount : %d \n", p_HostOutputAttributeMapping->i_MappingCount);
			fprintf(fp_RightLog, "[GpuJoinKernel] AttributeMapCount : %d \n", p_HostOutputAttributeMapping->i_MappingCount);
			for(int c=0; c<p_HostOutputAttributeMapping->i_MappingCount; ++c)
			{
				fprintf(fp_LeftLog, "[GpuJoinKernel] Map : Form [Stream=%d, Attrib=%d] To [Attrib=%d] \n",
						p_HostOutputAttributeMapping->p_Mappings[c].from[AttributeMapping::STREAM_INDEX],
						p_HostOutputAttributeMapping->p_Mappings[c].from[AttributeMapping::ATTRIBUTE_INDEX],
						p_HostOutputAttributeMapping->p_Mappings[c].to);

				fprintf(fp_RightLog, "[GpuJoinKernel] Map : Form [Stream=%d, Attrib=%d] To [Attrib=%d] \n",
						p_HostOutputAttributeMapping->p_Mappings[c].from[AttributeMapping::STREAM_INDEX],
						p_HostOutputAttributeMapping->p_Mappings[c].from[AttributeMapping::ATTRIBUTE_INDEX],
						p_HostOutputAttributeMapping->p_Mappings[c].to);
			}

			CUDA_CHECK_RETURN(hipMalloc(
					(void**) &p_DeviceOutputAttributeMapping,
					sizeof(AttributeMappings)));

			AttributeMappings * pHostMappings = (AttributeMappings*) malloc(sizeof(AttributeMappings));
			pHostMappings->i_MappingCount = p_HostOutputAttributeMapping->i_MappingCount;
			pHostMappings->p_Mappings = NULL;

			CUDA_CHECK_RETURN(hipMalloc(
					(void**) &pHostMappings->p_Mappings,
					sizeof(AttributeMapping) * p_HostOutputAttributeMapping->i_MappingCount));

			CUDA_CHECK_RETURN(hipMemcpy(
					pHostMappings->p_Mappings,
					p_HostOutputAttributeMapping->p_Mappings,
					sizeof(AttributeMapping) * p_HostOutputAttributeMapping->i_MappingCount,
					hipMemcpyHostToDevice));

			CUDA_CHECK_RETURN(hipMemcpy(
					p_DeviceOutputAttributeMapping,
					pHostMappings,
					sizeof(AttributeMappings),
					hipMemcpyHostToDevice));

			CUDA_CHECK_RETURN(hipPeekAtLastError());
			CUDA_CHECK_RETURN(hipDeviceSynchronize());

			free(pHostMappings);
			pHostMappings = NULL;
		}

		if(p_JoinProcessor->GetThreadWorkSize() != 0)
		{
			i_LeftThreadWorkSize = p_JoinProcessor->GetThreadWorkSize();
			i_RightThreadWorkSize = p_JoinProcessor->GetThreadWorkSize();
		}

		if(i_LeftThreadWorkSize >= i_RightStreamWindowSize)
		{
			i_LeftThreadWorkSize = i_RightStreamWindowSize;
		}
		if(i_RightThreadWorkSize >= i_LeftStreamWindowSize)
		{
			i_RightThreadWorkSize = i_LeftStreamWindowSize;
		}

		i_LeftThreadWorkerCount = ceil((float)i_RightStreamWindowSize / i_LeftThreadWorkSize);
		i_RightThreadWorkerCount = ceil((float)i_LeftStreamWindowSize / i_RightThreadWorkSize);

		fprintf(fp_LeftLog, "[GpuJoinKernel] LeftThreadWorkSize=%d RightThreadWorkSize=%d\n", i_LeftThreadWorkSize, i_RightThreadWorkSize);
		fflush(fp_LeftLog);
		fprintf(fp_RightLog, "[GpuJoinKernel] LeftThreadWorkSize=%d RightThreadWorkSize=%d\n", i_LeftThreadWorkSize, i_RightThreadWorkSize);
		fflush(fp_RightLog);

		fprintf(fp_LeftLog, "[GpuJoinKernel] LeftThreadWorkCount=%d RightThreadWorkCount=%d\n", i_LeftThreadWorkerCount, i_RightThreadWorkerCount);
		fflush(fp_LeftLog);
		fprintf(fp_RightLog, "[GpuJoinKernel] LeftThreadWorkCount=%d RightThreadWorkCount=%d\n", i_LeftThreadWorkerCount, i_RightThreadWorkerCount);
		fflush(fp_RightLog);

		CUDA_CHECK_RETURN(hipMalloc((void**) &p_DeviceParametersLeft, sizeof(JoinKernelParameters)));
		JoinKernelParameters * pHostParameters = (JoinKernelParameters*) malloc(sizeof(JoinKernelParameters));

		pHostParameters->p_InputEventBuffer = p_LeftInputEventBuffer->GetDeviceEventBuffer();
		pHostParameters->p_InputMetaEvent = p_LeftInputEventBuffer->GetDeviceMetaEvent();
		pHostParameters->p_EventWindowBuffer = p_LeftWindowEventBuffer->GetDeviceEventBuffer();
		pHostParameters->i_WindowLength = i_LeftStreamWindowSize;
		pHostParameters->p_OtherStreamMetaEvent = p_RightInputEventBuffer->GetDeviceMetaEvent();
		pHostParameters->p_OtherEventWindowBuffer = p_RightWindowEventBuffer->GetReadOnlyDeviceEventBuffer();
		pHostParameters->i_OtherWindowLength = i_RightStreamWindowSize;
		pHostParameters->p_OnCompareFilter = p_DeviceOnCompareFilter;
		pHostParameters->i_WithInTime = p_JoinProcessor->GetWithInTimeMilliSeconds();
		pHostParameters->p_OutputStreamMetaEvent = p_LeftResultEventBuffer->GetDeviceMetaEvent();
		pHostParameters->p_ResultsBuffer = p_LeftResultEventBuffer->GetDeviceEventBuffer();
		pHostParameters->p_OutputAttribMappings = p_DeviceOutputAttributeMapping;
		pHostParameters->i_EventsPerBlock = i_ThreadBlockSize;
		pHostParameters->i_WorkSize = i_LeftThreadWorkSize;

		CUDA_CHECK_RETURN(hipMemcpy(
				p_DeviceParametersLeft,
				pHostParameters,
				sizeof(JoinKernelParameters),
				hipMemcpyHostToDevice));

		free(pHostParameters);
		pHostParameters = NULL;

		CUDA_CHECK_RETURN(hipMalloc((void**) &p_DeviceParametersRight, sizeof(JoinKernelParameters)));
		pHostParameters = (JoinKernelParameters*) malloc(sizeof(JoinKernelParameters));

		pHostParameters->p_InputEventBuffer = p_RightInputEventBuffer->GetDeviceEventBuffer();
		pHostParameters->p_InputMetaEvent = p_RightInputEventBuffer->GetDeviceMetaEvent();
		pHostParameters->p_EventWindowBuffer = p_RightWindowEventBuffer->GetDeviceEventBuffer();
		pHostParameters->i_WindowLength = i_RightStreamWindowSize;
		pHostParameters->p_OtherStreamMetaEvent = p_LeftInputEventBuffer->GetDeviceMetaEvent();
		pHostParameters->p_OtherEventWindowBuffer = p_LeftWindowEventBuffer->GetReadOnlyDeviceEventBuffer();
		pHostParameters->i_OtherWindowLength = i_LeftStreamWindowSize;
		pHostParameters->p_OnCompareFilter = p_DeviceOnCompareFilter;
		pHostParameters->i_WithInTime = p_JoinProcessor->GetWithInTimeMilliSeconds();
		pHostParameters->p_OutputStreamMetaEvent = p_RightResultEventBuffer->GetDeviceMetaEvent();
		pHostParameters->p_ResultsBuffer = p_RightResultEventBuffer->GetDeviceEventBuffer();
		pHostParameters->p_OutputAttribMappings = p_DeviceOutputAttributeMapping;
		pHostParameters->i_EventsPerBlock = i_ThreadBlockSize;
		pHostParameters->i_WorkSize = i_RightThreadWorkSize;

		CUDA_CHECK_RETURN(hipMemcpy(
				p_DeviceParametersRight,
				pHostParameters,
				sizeof(JoinKernelParameters),
				hipMemcpyHostToDevice));

		free(pHostParameters);
		pHostParameters = NULL;

		fprintf(fp_LeftLog, "[GpuJoinKernel] Initialization complete\n");
		fflush(fp_LeftLog);
		fprintf(fp_RightLog, "[GpuJoinKernel] Initialization complete\n");
		fflush(fp_RightLog);
	}

	return true;
}

void GpuJoinKernel::Process(int _iStreamIndex, int & _iNumEvents)
{
	if(_iStreamIndex == 0)
	{
		ProcessLeftStream(_iStreamIndex, _iNumEvents);
	}
	else if(_iStreamIndex == 1)
	{
		ProcessRightStream(_iStreamIndex, _iNumEvents);
	}
}

void GpuJoinKernel::ProcessLeftStream(int _iStreamIndex, int & _iNumEvents)
{
#if GPU_DEBUG >= GPU_DEBUG_LEVEL_DEBUG
	fprintf(fp_LeftLog, "[GpuJoinKernel] ProcessLeftStream : StreamIndex=%d EventCount=%d\n", _iStreamIndex, _iNumEvents);
	GpuUtils::PrintThreadInfo("GpuJoinKernel::ProcessLeftStream", fp_LeftLog);
	fflush(fp_LeftLog);
#endif

	if(!b_LeftDeviceSet)
	{
		GpuCudaHelper::SelectDevice(i_DeviceId, "GpuJoinKernel::Left", fp_LeftLog);
		b_LeftDeviceSet = true;
	}

#ifdef KERNEL_TIME
	sdkStartTimer(&p_StopWatch);
#endif

	if(b_LeftFirstKernel)
	{
		p_LeftInputEventBuffer->CopyToDevice(true);
	}

	// call entry kernel
	int numBlocksX = ceil((float)_iNumEvents * i_LeftThreadWorkerCount / (float)i_ThreadBlockSize);
	int numBlocksY = 1;
	dim3 numBlocks = dim3(numBlocksX, numBlocksY);
	dim3 numThreads = dim3(i_ThreadBlockSize, 1);

#if GPU_DEBUG >= GPU_DEBUG_LEVEL_INFO
	fprintf(fp_LeftLog, "[GpuJoinKernel] ProcessLeftStream : Invoke kernel Blocks(%d,%d) Threads(%d,%d)\n", numBlocksX, numBlocksY, i_ThreadBlockSize, 1);
	fprintf(fp_LeftLog, "[GpuJoinKernel] ProcessLeftStream : NumEvents=%d LeftWindow=(%d/%d) RightWindow=(%d/%d) WithIn=%llu\n",
			_iNumEvents, p_LeftWindowEventBuffer->GetRemainingCount(), i_LeftStreamWindowSize, p_RightWindowEventBuffer->GetRemainingCount(),
			i_RightStreamWindowSize, p_JoinProcessor->GetWithInTimeMilliSeconds());
	fflush(fp_LeftLog);
#endif

#if GPU_DEBUG >= GPU_DEBUG_LEVEL_TRACE
	GpuUtils::PrintByteBuffer(p_LeftInputEventBuffer->GetHostEventBuffer(), _iNumEvents,
			p_LeftInputEventBuffer->GetHostMetaEvent(), "GpuJoinKernel:LeftInputBuffer", fp_LeftLog);

	p_LeftWindowEventBuffer->CopyToHost(false);
	GpuUtils::PrintByteBuffer(p_LeftWindowEventBuffer->GetHostEventBuffer(), (i_LeftStreamWindowSize - p_LeftWindowEventBuffer->GetRemainingCount()),
			p_LeftWindowEventBuffer->GetHostMetaEvent(), "GpuJoinKernel:LeftWindowBuffer", fp_LeftLog);

	p_RightWindowEventBuffer->CopyToHost(false);
	GpuUtils::PrintByteBuffer(p_RightWindowEventBuffer->GetHostEventBuffer(), (i_RightStreamWindowSize - p_RightWindowEventBuffer->GetRemainingCount()),
			p_RightWindowEventBuffer->GetHostMetaEvent(), "GpuJoinKernel:RightWindowBuffer", fp_LeftLog);

	fflush(fp_LeftLog);
#endif

//	char               * _pInputEventBuffer,         // input events buffer
//	GpuKernelMetaEvent * _pInputMetaEvent,           // Meta event for input events
//	int                  _iInputNumberOfEvents,      // Number of events in input buffer
//	char               * _pEventWindowBuffer,        // Event window buffer of this stream
//	int                  _iWindowLength,             // Length of current events window
//	int                  _iRemainingCount,           // Remaining free slots in Window buffer
//	GpuKernelMetaEvent * _pOtherStreamMetaEvent,     // Meta event for other stream
//	char               * _pOtherEventWindowBuffer,   // Event window buffer of other stream
//	int                  _iOtherWindowLength,        // Length of current events window of other stream
//	int                  _iOtherRemainingCount,      // Remaining free slots in Window buffer of other stream
//	GpuKernelFilter    * _pOnCompareFilter,          // OnCompare filter buffer - pre-copied at initialization
//	int                  _iWithInTime,               // WithIn time in milliseconds
//	GpuKernelMetaEvent * _pOutputStreamMetaEvent,    // Meta event for output stream
//	char               * _pResultsBuffer,            // Resulting events buffer for this stream
//	AttributeMappings  * _pOutputAttribMappings,     // Output event attribute mappings
//	int                  _iEventsPerBlock            // number of events allocated per block

	if(p_JoinProcessor->GetLeftTrigger())
	{

		if(p_JoinProcessor->GetCurrentOn() && p_JoinProcessor->GetExpiredOn())
		{
			ProcessEventsJoinLeftTriggerAllOn<<<numBlocks, numThreads>>>(
					p_LeftInputEventBuffer->GetDeviceEventBuffer(),
					p_LeftInputEventBuffer->GetDeviceMetaEvent(),
					_iNumEvents,
					p_LeftWindowEventBuffer->GetDeviceEventBuffer(),
					i_LeftStreamWindowSize,
					p_LeftWindowEventBuffer->GetRemainingCount(),
					p_RightInputEventBuffer->GetDeviceMetaEvent(),
					p_RightWindowEventBuffer->GetReadOnlyDeviceEventBuffer(),
					i_RightStreamWindowSize,
					p_RightWindowEventBuffer->GetRemainingCount(),
					p_DeviceOnCompareFilter,
					p_JoinProcessor->GetWithInTimeMilliSeconds(),
					p_LeftResultEventBuffer->GetDeviceMetaEvent(),
					p_LeftResultEventBuffer->GetDeviceEventBuffer(),
					p_DeviceOutputAttributeMapping,
					i_ThreadBlockSize
			);
		}
		else if(p_JoinProcessor->GetCurrentOn())
		{
			int iSharedSize = (i_ThreadBlockSize * p_LeftInputEventBuffer->GetHostMetaEvent()->i_SizeOfEventInBytes / i_LeftThreadWorkerCount);

			ProcessEventsJoinLeftTriggerCurrentOn<<<numBlocks, numThreads, iSharedSize>>>(
					p_DeviceParametersLeft,
					_iNumEvents,
					p_LeftWindowEventBuffer->GetRemainingCount(),
					p_RightWindowEventBuffer->GetRemainingCount()
			);
		}
		else if(p_JoinProcessor->GetExpiredOn())
		{
			ProcessEventsJoinLeftTriggerExpiredOn<<<numBlocks, numThreads>>>(
					p_LeftInputEventBuffer->GetDeviceEventBuffer(),
					p_LeftInputEventBuffer->GetDeviceMetaEvent(),
					_iNumEvents,
					p_LeftWindowEventBuffer->GetDeviceEventBuffer(),
					i_LeftStreamWindowSize,
					p_LeftWindowEventBuffer->GetRemainingCount(),
					p_RightInputEventBuffer->GetDeviceMetaEvent(),
					p_RightWindowEventBuffer->GetReadOnlyDeviceEventBuffer(),
					i_RightStreamWindowSize,
					p_RightWindowEventBuffer->GetRemainingCount(),
					p_DeviceOnCompareFilter,
					p_JoinProcessor->GetWithInTimeMilliSeconds(),
					p_LeftResultEventBuffer->GetDeviceMetaEvent(),
					p_LeftResultEventBuffer->GetDeviceEventBuffer(),
					p_DeviceOutputAttributeMapping,
					i_ThreadBlockSize
			);
		}

	}

	if(b_LastKernel)
	{
		p_LeftResultEventBuffer->CopyToHost(true);
#if GPU_DEBUG >= GPU_DEBUG_LEVEL_DEBUG
	fprintf(fp_LeftLog, "[GpuJoinKernel] Results copied \n");
	fflush(fp_LeftLog);
#endif
	}

	numBlocksX = ceil((float)_iNumEvents / (float)i_ThreadBlockSize);
	numBlocks = dim3(numBlocksX, numBlocksY);

	// we need to synchronize processing of JoinKernel as only one batch of events can be there at a time
//	pthread_mutex_lock(&mtx_Lock);

//	char               * _pInputEventBuffer,     // original input events buffer
//	int                  _iNumberOfEvents,       // Number of events in input buffer (matched + not matched)
//	char               * _pEventWindowBuffer,    // Event window buffer
//	int                  _iWindowLength,         // Length of current events window
//	int                  _iRemainingCount,       // Remaining free slots in Window buffer
//	int                  _iMaxEventCount,        // used for setting results array
//	int                  _iSizeOfEvent,          // Size of an event
//	int                  _iEventsPerBlock        // number of events allocated per block

	JoinSetWindowState<<<numBlocks, numThreads>>>(
			p_LeftInputEventBuffer->GetDeviceEventBuffer(),
			_iNumEvents,
			p_LeftWindowEventBuffer->GetDeviceEventBuffer(),
			i_LeftStreamWindowSize,
			p_LeftWindowEventBuffer->GetRemainingCount(),
			p_LeftInputEventBuffer->GetMaxEventCount(),
			p_LeftInputEventBuffer->GetHostMetaEvent()->i_SizeOfEventInBytes,
			i_ThreadBlockSize
	);

	CUDA_CHECK_RETURN(hipPeekAtLastError());
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	p_LeftWindowEventBuffer->Sync(_iNumEvents, true);

#if GPU_DEBUG >= GPU_DEBUG_LEVEL_INFO
	fprintf(fp_LeftLog, "[GpuJoinKernel] Kernel complete \n");
	fflush(fp_LeftLog);
#endif



#ifdef KERNEL_TIME
	sdkStopTimer(&p_StopWatch);
	float fElapsed = sdkGetTimerValue(&p_StopWatch);
	fprintf(fp_LeftLog, "[GpuJoinKernel] Stats : Elapsed=%f ms\n", fElapsed);
	fflush(fp_LeftLog);
	lst_ElapsedTimes.push_back(fElapsed);
	sdkResetTimer(&p_StopWatch);
#endif


//	if(_iNumEvents > i_LeftRemainingCount)
//	{
//		i_LeftRemainingCount = 0;
//	}
//	else
//	{
//		i_LeftRemainingCount -= _iNumEvents;
//	}

//	pthread_mutex_unlock(&mtx_Lock);

	if(!p_JoinProcessor->GetLeftTrigger())
	{
		_iNumEvents = 0;
	}
	else
	{
		_iNumEvents = _iNumEvents * i_LeftNumEventPerSegment;
	}


#if GPU_DEBUG >= GPU_DEBUG_LEVEL_DEBUG
	GpuUtils::PrintByteBuffer(p_LeftResultEventBuffer->GetHostEventBuffer(), _iNumEvents,
			p_LeftResultEventBuffer->GetHostMetaEvent(), "GpuJoinKernel:LeftResultEventBuffer", fp_LeftLog);
	fflush(fp_LeftLog);
#endif

#if GPU_DEBUG >= GPU_DEBUG_LEVEL_TRACE
	p_LeftWindowEventBuffer->CopyToHost(true);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	GpuUtils::PrintByteBuffer(p_LeftWindowEventBuffer->GetHostEventBuffer(), (i_LeftStreamWindowSize - p_LeftWindowEventBuffer->GetRemainingCount()),
			p_LeftWindowEventBuffer->GetHostMetaEvent(), "GpuJoinKernel:LeftWindowBuffer", fp_LeftLog);
	fflush(fp_LeftLog);
#endif
}

void GpuJoinKernel::ProcessRightStream(int _iStreamIndex, int & _iNumEvents)
{
#if GPU_DEBUG >= GPU_DEBUG_LEVEL_DEBUG
	fprintf(fp_RightLog, "[GpuJoinKernel] ProcessRightStream : StreamIndex=%d EventCount=%d\n", _iStreamIndex, _iNumEvents);
	GpuUtils::PrintThreadInfo("GpuJoinKernel::ProcessRightStream", fp_RightLog);
	fflush(fp_RightLog);
#endif

	if(!b_RightDeviceSet)
	{
		GpuCudaHelper::SelectDevice(i_DeviceId, "GpuJoinKernel::Right", fp_RightLog);
		b_RightDeviceSet = true;
	}

#ifdef KERNEL_TIME
	sdkStartTimer(&p_StopWatch);
#endif

	if(b_RightFirstKernel)
	{
		p_RightInputEventBuffer->CopyToDevice(true);
	}

	// call entry kernel
	int numBlocksX = ceil((float)_iNumEvents * i_RightThreadWorkerCount / (float)i_ThreadBlockSize);
	int numBlocksY = 1;
	dim3 numBlocks = dim3(numBlocksX, numBlocksY);
	dim3 numThreads = dim3(i_ThreadBlockSize, 1);

#if GPU_DEBUG >= GPU_DEBUG_LEVEL_INFO
	fprintf(fp_RightLog, "[GpuJoinKernel] ProcessRightStream : Invoke kernel Blocks(%d,%d) Threads(%d,%d)\n", numBlocksX, numBlocksY, i_ThreadBlockSize, 1);
	fprintf(fp_RightLog, "[GpuJoinKernel] ProcessRightStream : NumEvents=%d LeftWindow=(%d/%d) RightWindow=(%d/%d) WithIn=%llu\n",
			_iNumEvents, p_LeftWindowEventBuffer->GetRemainingCount(), i_LeftStreamWindowSize, p_RightWindowEventBuffer->GetRemainingCount(),
			i_RightStreamWindowSize, p_JoinProcessor->GetWithInTimeMilliSeconds());
	fflush(fp_RightLog);
#endif

#if GPU_DEBUG >= GPU_DEBUG_LEVEL_TRACE
	GpuUtils::PrintByteBuffer(p_RightInputEventBuffer->GetHostEventBuffer(), _iNumEvents,
			p_RightInputEventBuffer->GetHostMetaEvent(), "GpuJoinKernel:RightInputBuffer", fp_RightLog);

	p_LeftWindowEventBuffer->CopyToHost(false);
	GpuUtils::PrintByteBuffer(p_LeftWindowEventBuffer->GetHostEventBuffer(), (i_LeftStreamWindowSize - p_LeftWindowEventBuffer->GetRemainingCount()),
			p_LeftWindowEventBuffer->GetHostMetaEvent(), "GpuJoinKernel:LeftWindowBuffer", fp_RightLog);

	p_RightWindowEventBuffer->CopyToHost(false);
	GpuUtils::PrintByteBuffer(p_RightWindowEventBuffer->GetHostEventBuffer(), (i_RightStreamWindowSize - p_RightWindowEventBuffer->GetRemainingCount()),
			p_RightWindowEventBuffer->GetHostMetaEvent(), "GpuJoinKernel:RightWindowBuffer", fp_RightLog);

	fflush(fp_RightLog);
#endif

//	char               * _pInputEventBuffer,         // input events buffer
//	GpuKernelMetaEvent * _pInputMetaEvent,           // Meta event for input events
//	int                  _iInputNumberOfEvents,      // Number of events in input buffer
//	char               * _pEventWindowBuffer,        // Event window buffer of this stream
//	int                  _iWindowLength,             // Length of current events window
//	int                  _iRemainingCount,           // Remaining free slots in Window buffer
//	GpuKernelMetaEvent * _pOtherStreamMetaEvent,     // Meta event for other stream
//	char               * _pOtherEventWindowBuffer,   // Event window buffer of other stream
//	int                  _iOtherWindowLength,        // Length of current events window of other stream
//	int                  _iOtherRemainingCount,      // Remaining free slots in Window buffer of other stream
//	GpuKernelFilter    * _pOnCompareFilter,          // OnCompare filter buffer - pre-copied at initialization
//	int                  _iWithInTime,               // WithIn time in milliseconds
//	GpuKernelMetaEvent * _pOutputStreamMetaEvent,    // Meta event for output stream
//	char               * _pResultsBuffer,            // Resulting events buffer for this stream
//	AttributeMappings  * _pOutputAttribMappings,     // Output event attribute mappings
//	int                  _iEventsPerBlock            // number of events allocated per block

	if(p_JoinProcessor->GetRightTrigger())
	{
		if(p_JoinProcessor->GetCurrentOn() && p_JoinProcessor->GetExpiredOn())
		{
			ProcessEventsJoinRightTriggerAllOn<<<numBlocks, numThreads>>>(
					p_RightInputEventBuffer->GetDeviceEventBuffer(),
					p_RightInputEventBuffer->GetDeviceMetaEvent(),
					_iNumEvents,
					p_RightWindowEventBuffer->GetDeviceEventBuffer(),
					i_RightStreamWindowSize,
					p_RightWindowEventBuffer->GetRemainingCount(),
					p_LeftInputEventBuffer->GetDeviceMetaEvent(),
					p_LeftWindowEventBuffer->GetReadOnlyDeviceEventBuffer(),
					i_LeftStreamWindowSize,
					p_LeftWindowEventBuffer->GetRemainingCount(),
					p_DeviceOnCompareFilter,
					p_JoinProcessor->GetWithInTimeMilliSeconds(),
					p_RightResultEventBuffer->GetDeviceMetaEvent(),
					p_RightResultEventBuffer->GetDeviceEventBuffer(),
					p_DeviceOutputAttributeMapping,
					i_ThreadBlockSize
			);
		}
		else if(p_JoinProcessor->GetCurrentOn())
		{
			int iSharedSize = (i_ThreadBlockSize * p_RightInputEventBuffer->GetHostMetaEvent()->i_SizeOfEventInBytes / i_RightThreadWorkerCount);

			ProcessEventsJoinRightTriggerCurrentOn<<<numBlocks, numThreads, iSharedSize>>>(
					p_DeviceParametersRight,
					_iNumEvents,
					p_RightWindowEventBuffer->GetRemainingCount(),
					p_LeftWindowEventBuffer->GetRemainingCount()
			);
		}
		else if(p_JoinProcessor->GetExpiredOn())
		{
			ProcessEventsJoinRightTriggerExpireOn<<<numBlocks, numThreads>>>(
					p_RightInputEventBuffer->GetDeviceEventBuffer(),
					p_RightInputEventBuffer->GetDeviceMetaEvent(),
					_iNumEvents,
					p_RightWindowEventBuffer->GetDeviceEventBuffer(),
					i_RightStreamWindowSize,
					p_RightWindowEventBuffer->GetRemainingCount(),
					p_LeftInputEventBuffer->GetDeviceMetaEvent(),
					p_LeftWindowEventBuffer->GetReadOnlyDeviceEventBuffer(),
					i_LeftStreamWindowSize,
					p_LeftWindowEventBuffer->GetRemainingCount(),
					p_DeviceOnCompareFilter,
					p_JoinProcessor->GetWithInTimeMilliSeconds(),
					p_RightResultEventBuffer->GetDeviceMetaEvent(),
					p_RightResultEventBuffer->GetDeviceEventBuffer(),
					p_DeviceOutputAttributeMapping,
					i_ThreadBlockSize
			);
		}

	}

	if(b_LastKernel)
	{
		p_RightResultEventBuffer->CopyToHost(true);
#if GPU_DEBUG >= GPU_DEBUG_LEVEL_DEBUG
	fprintf(fp_RightLog, "[GpuJoinKernel] Results copied \n");
	fflush(fp_RightLog);
#endif
	}

	numBlocksX = ceil((float)_iNumEvents / (float)i_ThreadBlockSize);
	numBlocks = dim3(numBlocksX, numBlocksY);

	// we need to synchronize processing of JoinKernel as only one batch of events can be there at a time
//	pthread_mutex_lock(&mtx_Lock);

//	char               * _pInputEventBuffer,     // original input events buffer
//	int                  _iNumberOfEvents,       // Number of events in input buffer (matched + not matched)
//	char               * _pEventWindowBuffer,    // Event window buffer
//	int                  _iWindowLength,         // Length of current events window
//	int                  _iRemainingCount,       // Remaining free slots in Window buffer
//	int                  _iMaxEventCount,        // used for setting results array
//	int                  _iSizeOfEvent,          // Size of an event
//	int                  _iEventsPerBlock        // number of events allocated per block

	JoinSetWindowState<<<numBlocks, numThreads>>>(
			p_RightInputEventBuffer->GetDeviceEventBuffer(),
			_iNumEvents,
			p_RightWindowEventBuffer->GetDeviceEventBuffer(),
			i_RightStreamWindowSize,
			p_RightWindowEventBuffer->GetRemainingCount(),
			p_RightInputEventBuffer->GetMaxEventCount(),
			p_RightInputEventBuffer->GetHostMetaEvent()->i_SizeOfEventInBytes,
			i_ThreadBlockSize
	);

	CUDA_CHECK_RETURN(hipPeekAtLastError());
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	p_RightWindowEventBuffer->Sync(_iNumEvents, true);

#if GPU_DEBUG >= GPU_DEBUG_LEVEL_INFO
	fprintf(fp_RightLog, "[GpuJoinKernel] Kernel complete \n");
	fflush(fp_RightLog);
#endif



#ifdef KERNEL_TIME
	sdkStopTimer(&p_StopWatch);
	float fElapsed = sdkGetTimerValue(&p_StopWatch);
	fprintf(fp_RightLog, "[GpuJoinKernel] Stats : Elapsed=%f ms\n", fElapsed);
	fflush(fp_RightLog);
	lst_ElapsedTimes.push_back(fElapsed);
	sdkResetTimer(&p_StopWatch);
#endif


//	if(_iNumEvents > i_RightRemainingCount)
//	{
//		i_RightRemainingCount = 0;
//	}
//	else
//	{
//		i_RightRemainingCount -= _iNumEvents;
//	}

//	pthread_mutex_unlock(&mtx_Lock);

	if(!p_JoinProcessor->GetRightTrigger())
	{
		_iNumEvents = 0;
	}
	else
	{
		_iNumEvents = _iNumEvents * i_RightNumEventPerSegment;
	}


#if GPU_DEBUG >= GPU_DEBUG_LEVEL_DEBUG
	GpuUtils::PrintByteBuffer(p_RightResultEventBuffer->GetHostEventBuffer(), _iNumEvents,
			p_RightResultEventBuffer->GetHostMetaEvent(), "GpuJoinKernel:RightResultEventBuffer", fp_RightLog);
	fflush(fp_RightLog);
#endif

#if GPU_DEBUG >= GPU_DEBUG_LEVEL_TRACE
	p_RightWindowEventBuffer->CopyToHost(true);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	GpuUtils::PrintByteBuffer(p_RightWindowEventBuffer->GetHostEventBuffer(), (i_RightStreamWindowSize - p_RightWindowEventBuffer->GetRemainingCount()),
			p_RightWindowEventBuffer->GetHostMetaEvent(), "GpuJoinKernel:RightWindowBuffer", fp_RightLog);
	fflush(fp_RightLog);
#endif
}

char * GpuJoinKernel::GetResultEventBuffer()
{
	return NULL;
}

int GpuJoinKernel::GetResultEventBufferSize()
{
	return 0;
}

char * GpuJoinKernel::GetLeftResultEventBuffer()
{
	return p_LeftResultEventBuffer->GetHostEventBuffer();
}

int GpuJoinKernel::GetLeftResultEventBufferSize()
{
	return p_LeftResultEventBuffer->GetEventBufferSizeInBytes();
}

char * GpuJoinKernel::GetRightResultEventBuffer()
{
	return p_RightResultEventBuffer->GetHostEventBuffer();
}

int GpuJoinKernel::GetRightResultEventBufferSize()
{
	return p_RightResultEventBuffer->GetEventBufferSizeInBytes();
}

}

#endif
